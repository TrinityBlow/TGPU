#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}
__global__ void matDet(double *d_matA, double *detM){ 
    
	int global_id =  blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    __shared__ double s_mat[sizeof(double)*64];
    __shared__ double s_detAux[16];
    int offset = (threadIdx.y * blockDim.x + threadIdx.x)*16; 

    if ((threadIdx.y * blockDim.x + threadIdx.x) < 64){
        s_mat[(threadIdx.y * blockDim.x + threadIdx.x)]=d_matA[global_id];
        if(threadIdx.y * blockDim.x + threadIdx.x < 16){
            s_detAux[threadIdx.y * blockDim.x + threadIdx.x] = 0;
        }
        __syncthreads();
 
        if(threadIdx.y * blockDim.x + threadIdx.x < 4){
          //  printf("globalId:%d|%d|%d|%d|%d\n",global_id,(threadIdx.y * blockDim.x + threadIdx.x)*4,(threadIdx.y * blockDim.x + threadIdx.x)*4+1,(threadIdx.y * blockDim.x + threadIdx.x)*4+2,(threadIdx.y * blockDim.x + threadIdx.x)*4+3);
            s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4] +=  s_mat[offset] * ( (s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+13])+(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+13]))   + (-1*(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+15])) );
    //        __syncthreads();       
           
            s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+1] +=  (-1*s_mat[offset+1]) * ( (s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+15])) );    
     //       __syncthreads();

            s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+2] +=  s_mat[offset+2] * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+15])+(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+15])) );        
     //       __syncthreads();
      
            s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+3] +=  (-1*s_mat[offset+3]) * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+14])+(s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+12])+(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+14])) );            
       //     __syncthreads();
          
            detM[blockIdx.x*4 + (threadIdx.y * blockDim.x + threadIdx.x)] = s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+1] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+2] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+3]; 
  //          __syncthreads();
          
        }
    }
}

__global__ void vecMult2(double *d_matA,unsigned long n,int iteraciones){      
	int global_id =  blockIdx.x *blockDim.x + threadIdx.x;
    __shared__ double s_mat[32];
    unsigned int i;
        /*
        for( i = 1; i <= 2;  i++) {
            if(( threadIdx.y * blockDim.x + threadIdx.x )< (int)(64 >> i)){
                s_mat[(threadIdx.y * blockDim.x + threadIdx.x)] += s_mat[((threadIdx.y * blockDim.x + threadIdx.x ) + (64 >> i))];
            }
            __syncthreads();
        }
        */
    for(i = 1 ; i <= iteraciones ; i++){
        if ( global_id < ( n / (1 << (i - 1) ))){
            s_mat[threadIdx.x]=d_matA[global_id];
       //     printf("global:%d||%.2lf||\n",global_id,s_mat[threadIdx.x]);
        }
         __syncthreads();

        if ( global_id < ( n / (1 << (i - 1) ))){
            if (threadIdx.x < 16){
                printf("global:%d||%.2lf||%.2lf||\n",global_id,s_mat[threadIdx.x],s_mat[threadIdx.x + 16]);
                s_mat[threadIdx.x] += s_mat[threadIdx.x + 16];           
            }
        }

        __syncthreads();

        if ( global_id < ( n / (1 << (i - 1) ))){
            if (threadIdx.x < 16){
    //          printf("global:%d||%.2lf||\n",global_id,s_mat[threadIdx.x]);
                d_matA[blockIdx.x * 16 + threadIdx.x] = s_mat[threadIdx.x];      
            }
        }
        __syncthreads();
            if(global_id == 0){
                printf("-------------------------------------------\n");

            }
    }
}





int main(int argc, char *argv[]){


    if (argc != 2){
        printf("Falta argumento: N\n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned long N = atoi (argv[1]);
    unsigned long CUDA_BLK = 32,GRID_BLK;
    unsigned long numBytes = sizeof(double)*4*4;
    double *matrices,*d_matrices,*d_detM,*detM,timetick;
    unsigned long i,j;
    int iteraciones;


    matrices = (double *)malloc(numBytes*N);
    detM = (double *)malloc(sizeof(double)*N);
    for (i = 0; i < 4*4*N; i++){
        matrices[i] = 1;
    }

    for (i = 0; i < N; i++){
        detM[i] = 0;
    }
    matrices[2] = 220;
    matrices[13] = 220;
    matrices[7] = 6;
    matrices[14] = 6;
    //comment

    hipMalloc((void **) &d_matrices, numBytes*N);
    hipMalloc((void **) &d_detM, sizeof(double)*N);

    dim3 dimBlock(CUDA_BLK);
    dim3 dimGrid(N/4);
    
    timetick = dwalltime();

    iteraciones = log(N) / log(2);

    hipMemcpy(d_matrices, matrices, numBytes*N, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_detM, detM, sizeof(double)*N, hipMemcpyHostToDevice); // CPU -> GPU
  //  matDet<<<dimGrid, dimBlock>>>(d_matrices,d_detM);
 //   hipDeviceSynchronize();
    dim3 dimGrid2(4*4*N/CUDA_BLK );
    vecMult2<<<dimGrid2, dimBlock>>>(d_matrices,(4*4*N),iteraciones);
    hipDeviceSynchronize();
    hipMemcpy(matrices, d_matrices, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    hipMemcpy(detM, d_detM, sizeof(double)*N, hipMemcpyDeviceToHost); // GPU -> CPU


	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n",error);
    
    printf("%.2lf|\n",detM[0]);

    for(i=0; i < 4; i++){
        for(j=0; j < 4; j++){
            printf("%.2lf|",matrices[i*4+j]);
        }
        printf("\n");
    }


    hipFree(d_matrices);
    hipFree(d_detM);
    free(matrices);
    free(detM);
    return 0;

}