#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

#define BASETYPE float

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

__global__ void matDet(BASETYPE *d_matA, BASETYPE *detM, int desp){ 
//	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int distA_id = blockIdx.x * blockDim.x * 16;
    extern __shared__ BASETYPE datos[];
    BASETYPE *s_mat = &datos[0];
    BASETYPE *s_detAux = &datos[desp];
    int offset = (threadIdx.x)*16; 

	s_mat[threadIdx.x]=d_matA[distA_id + threadIdx.x];
	s_mat[threadIdx.x + blockDim.x]=d_matA[distA_id + threadIdx.x + blockDim.x];
	s_mat[threadIdx.x + blockDim.x * 2]=d_matA[distA_id + threadIdx.x + blockDim.x * 2];
	s_mat[threadIdx.x + blockDim.x * 3]=d_matA[distA_id + threadIdx.x + blockDim.x * 3];
	s_mat[threadIdx.x + blockDim.x * 4]=d_matA[distA_id + threadIdx.x + blockDim.x * 4];
	s_mat[threadIdx.x + blockDim.x * 5]=d_matA[distA_id + threadIdx.x + blockDim.x * 5];
	s_mat[threadIdx.x + blockDim.x * 6]=d_matA[distA_id + threadIdx.x + blockDim.x * 6];
	s_mat[threadIdx.x + blockDim.x * 7]=d_matA[distA_id + threadIdx.x + blockDim.x * 7];
	s_mat[threadIdx.x + blockDim.x * 8]=d_matA[distA_id + threadIdx.x + blockDim.x * 8];
	s_mat[threadIdx.x + blockDim.x * 9]=d_matA[distA_id + threadIdx.x + blockDim.x * 9];
	s_mat[threadIdx.x + blockDim.x * 10]=d_matA[distA_id + threadIdx.x + blockDim.x * 10];
	s_mat[threadIdx.x + blockDim.x * 11]=d_matA[distA_id + threadIdx.x + blockDim.x * 11];
	s_mat[threadIdx.x + blockDim.x * 12]=d_matA[distA_id + threadIdx.x + blockDim.x * 12];
	s_mat[threadIdx.x + blockDim.x * 13]=d_matA[distA_id + threadIdx.x + blockDim.x * 13];
	s_mat[threadIdx.x + blockDim.x * 14]=d_matA[distA_id + threadIdx.x + blockDim.x * 14];
	s_mat[threadIdx.x + blockDim.x * 15]=d_matA[distA_id + threadIdx.x + blockDim.x * 15];
	__syncthreads();

    s_detAux[(threadIdx.x) * 4]=0;
    s_detAux[(threadIdx.x) * 4 + 1]=0;
    s_detAux[(threadIdx.x) * 4 + 2]=0;
    s_detAux[(threadIdx.x) * 4 + 3]=0;
    __syncthreads();

    //  printf("globalId:%d|%d|%d|%d|%d\n",global_id,(threadIdx.x)*4,(threadIdx.x)*4+1,(threadIdx.x)*4+2,(threadIdx.x)*4+3);
    s_detAux[(threadIdx.x)*4] +=  s_mat[offset] * ( (s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+13])+(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+13]))   + (-1*(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+15])) );

    s_detAux[(threadIdx.x)*4+1] +=  (-1*s_mat[offset+1]) * ( (s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+15])) );    

    s_detAux[(threadIdx.x)*4+2] +=  s_mat[offset+2] * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+15])+(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+15])) );        

    s_detAux[(threadIdx.x)*4+3] +=  (-1*s_mat[offset+3]) * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+14])+(s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+12])+(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+14])) );        
    detM[blockIdx.x * blockDim.x + (threadIdx.x)] = s_detAux[(threadIdx.x)*4] + s_detAux[(threadIdx.x)*4+1] + s_detAux[(threadIdx.x)*4+2] + s_detAux[(threadIdx.x)*4+3]; 
    __syncthreads();
        
}

__global__ void vecMult(BASETYPE *d_matA){     
//	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int distA_id = blockIdx.x * blockDim.x * 16;
    int distB_id = threadIdx.x * 16;
    int distC_id;
    extern __shared__ BASETYPE s_mat[];
    unsigned int j;

	s_mat[threadIdx.x]=d_matA[distA_id + threadIdx.x];
	s_mat[threadIdx.x + blockDim.x]=d_matA[distA_id + threadIdx.x + blockDim.x];
	s_mat[threadIdx.x + blockDim.x * 2]=d_matA[distA_id + threadIdx.x + blockDim.x * 2];
	s_mat[threadIdx.x + blockDim.x * 3]=d_matA[distA_id + threadIdx.x + blockDim.x * 3];
	s_mat[threadIdx.x + blockDim.x * 4]=d_matA[distA_id + threadIdx.x + blockDim.x * 4];
	s_mat[threadIdx.x + blockDim.x * 5]=d_matA[distA_id + threadIdx.x + blockDim.x * 5];
	s_mat[threadIdx.x + blockDim.x * 6]=d_matA[distA_id + threadIdx.x + blockDim.x * 6];
	s_mat[threadIdx.x + blockDim.x * 7]=d_matA[distA_id + threadIdx.x + blockDim.x * 7];
	s_mat[threadIdx.x + blockDim.x * 8]=d_matA[distA_id + threadIdx.x + blockDim.x * 8];
	s_mat[threadIdx.x + blockDim.x * 9]=d_matA[distA_id + threadIdx.x + blockDim.x * 9];
	s_mat[threadIdx.x + blockDim.x * 10]=d_matA[distA_id + threadIdx.x + blockDim.x * 10];
	s_mat[threadIdx.x + blockDim.x * 11]=d_matA[distA_id + threadIdx.x + blockDim.x * 11];
	s_mat[threadIdx.x + blockDim.x * 12]=d_matA[distA_id + threadIdx.x + blockDim.x * 12];
	s_mat[threadIdx.x + blockDim.x * 13]=d_matA[distA_id + threadIdx.x + blockDim.x * 13];
	s_mat[threadIdx.x + blockDim.x * 14]=d_matA[distA_id + threadIdx.x + blockDim.x * 14];
	s_mat[threadIdx.x + blockDim.x * 15]=d_matA[distA_id + threadIdx.x + blockDim.x * 15];
	__syncthreads();

	for( j = 1; j < blockDim.x; j *= 2 ){
	    if( threadIdx.x < blockDim.x / (j * 2)){
            distC_id = (blockDim.x / (j * 2)) * 16;
            s_mat[distB_id] += s_mat[(distB_id) + distC_id];
            s_mat[distB_id + 1] += s_mat[(distB_id + 1) + distC_id];
            s_mat[distB_id + 2] += s_mat[(distB_id + 2) + distC_id];
            s_mat[distB_id + 3] += s_mat[(distB_id + 3) + distC_id];
            s_mat[distB_id + 4] += s_mat[(distB_id + 4) + distC_id];
            s_mat[distB_id + 5] += s_mat[(distB_id + 5) + distC_id];
            s_mat[distB_id + 6] += s_mat[(distB_id + 6) + distC_id];
            s_mat[distB_id + 7] += s_mat[(distB_id + 7) + distC_id];
            s_mat[distB_id + 8] += s_mat[(distB_id + 8) + distC_id];
            s_mat[distB_id + 9] += s_mat[(distB_id + 9) + distC_id];
            s_mat[distB_id + 10] += s_mat[(distB_id + 10) + distC_id];
            s_mat[distB_id + 11] += s_mat[(distB_id + 11) + distC_id];
            s_mat[distB_id + 12] += s_mat[(distB_id + 12) + distC_id];
            s_mat[distB_id + 13] += s_mat[(distB_id + 13) + distC_id];
            s_mat[distB_id + 14] += s_mat[(distB_id + 14) + distC_id];
            s_mat[distB_id + 15] += s_mat[(distB_id + 15) + distC_id];
	        
	    }
	    __syncthreads();
	}


	if ((threadIdx.x) == 0){
        d_matA[(blockIdx.x * 16)] = s_mat[0];
        d_matA[(blockIdx.x * 16) + 1] = s_mat[1];
        d_matA[(blockIdx.x * 16) + 2] = s_mat[2];
        d_matA[(blockIdx.x * 16) + 3] = s_mat[3];
        d_matA[(blockIdx.x * 16) + 4] = s_mat[4];
        d_matA[(blockIdx.x * 16) + 5] = s_mat[5];
        d_matA[(blockIdx.x * 16) + 6] = s_mat[6];
        d_matA[(blockIdx.x * 16) + 7] = s_mat[7];
        d_matA[(blockIdx.x * 16) + 8] = s_mat[8];
        d_matA[(blockIdx.x * 16) + 9] = s_mat[9];
        d_matA[(blockIdx.x * 16) + 10] = s_mat[10];
        d_matA[(blockIdx.x * 16) + 11] = s_mat[11];
        d_matA[(blockIdx.x * 16) + 12] = s_mat[12];
        d_matA[(blockIdx.x * 16) + 13] = s_mat[13];
        d_matA[(blockIdx.x * 16) + 14] = s_mat[14];
        d_matA[(blockIdx.x * 16) + 15] = s_mat[15];
	}
}

__global__ void vecMult2(BASETYPE *d_matA,unsigned long n,int offset_m,int cant_m ){     
	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int distB_id = threadIdx.x * 16;
    extern __shared__ BASETYPE s_mat[];
    unsigned int j;

    if( global_id < n){

        s_mat[threadIdx.x]=d_matA[(offset_m * 16) + threadIdx.x];
        s_mat[threadIdx.x + n]=d_matA[(offset_m * 16) + threadIdx.x + n];
        s_mat[threadIdx.x + n * 2]=d_matA[(offset_m * 16) + threadIdx.x + n * 2];
        s_mat[threadIdx.x + n * 3]=d_matA[(offset_m * 16) + threadIdx.x + n * 3];
        s_mat[threadIdx.x + n * 4]=d_matA[(offset_m * 16) + threadIdx.x + n * 4];
        s_mat[threadIdx.x + n * 5]=d_matA[(offset_m * 16) + threadIdx.x + n * 5];
        s_mat[threadIdx.x + n * 6]=d_matA[(offset_m * 16) + threadIdx.x + n * 6];
        s_mat[threadIdx.x + n * 7]=d_matA[(offset_m * 16) + threadIdx.x + n * 7];
        s_mat[threadIdx.x + n * 8]=d_matA[(offset_m * 16) + threadIdx.x + n * 8];
        s_mat[threadIdx.x + n * 9]=d_matA[(offset_m * 16) + threadIdx.x + n * 9];
        s_mat[threadIdx.x + n * 10]=d_matA[(offset_m * 16) + threadIdx.x + n * 10];
        s_mat[threadIdx.x + n * 11]=d_matA[(offset_m * 16) + threadIdx.x + n * 11];
        s_mat[threadIdx.x + n * 12]=d_matA[(offset_m * 16) + threadIdx.x + n * 12];
        s_mat[threadIdx.x + n * 13]=d_matA[(offset_m * 16) + threadIdx.x + n * 13];
        s_mat[threadIdx.x + n * 14]=d_matA[(offset_m * 16) + threadIdx.x + n * 14];
        s_mat[threadIdx.x + n * 15]=d_matA[(offset_m * 16) + threadIdx.x + n * 15];
        __syncthreads();

        for( j = 1; j < cant_m; j *= 2 ){
            if( threadIdx.x < cant_m / (j * 2)){
                s_mat[distB_id] += s_mat[(distB_id) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 1] += s_mat[(distB_id + 1) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 2] += s_mat[(distB_id + 2) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 3] += s_mat[(distB_id + 3) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 4] += s_mat[(distB_id + 4) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 5] += s_mat[(distB_id + 5) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 6] += s_mat[(distB_id + 6) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 7] += s_mat[(distB_id + 7) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 8] += s_mat[(distB_id + 8) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 9] += s_mat[(distB_id + 9) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 10] += s_mat[(distB_id + 10) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 11] += s_mat[(distB_id + 11) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 12] += s_mat[(distB_id + 12) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 13] += s_mat[(distB_id + 13) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 14] += s_mat[(distB_id + 14) + (cant_m / (j * 2)) * 16];
                s_mat[distB_id + 15] += s_mat[(distB_id + 15) + (cant_m / (j * 2)) * 16];
                
            }
            __syncthreads();
        }


        if ((threadIdx.x) == 0){
            d_matA[(offset_m / blockDim.x) * 16 + (blockIdx.x * 16)] = s_mat[0];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 1] = s_mat[1];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 2] = s_mat[2];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 3] = s_mat[3];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 4] = s_mat[4];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 5] = s_mat[5];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 6] = s_mat[6];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 7] = s_mat[7];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 8] = s_mat[8];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 9] = s_mat[9];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 10] = s_mat[10];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 11] = s_mat[11];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 12] = s_mat[12];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 13] = s_mat[13];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 14] = s_mat[14];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 15] = s_mat[15];
        }
    }
}



__global__ void sumDet(BASETYPE *detM ){   
	//int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ BASETYPE s_vec[];
    
    unsigned int j;

	s_vec[threadIdx.x]=detM[ blockIdx.x * blockDim.x + threadIdx.x];
	__syncthreads();

	for( j = 1; j < blockDim.x; j *= 2 ){
	    if( threadIdx.x < blockDim.x / (j * 2)){
            s_vec[threadIdx.x] += s_vec[threadIdx.x + (blockDim.x / (j * 2))];
	    }
	    __syncthreads();
	}


	if ((threadIdx.x) == 0){
        detM[blockIdx.x] = s_vec[0];
	}


}

__global__ void sumDet2(BASETYPE *detM,unsigned long n,int offset_m,int cant_m ){     
	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ BASETYPE s_vec[];
    unsigned int j;

    if( global_id < n){

        s_vec[threadIdx.x]=detM[offset_m  + threadIdx.x];
        __syncthreads();

        for( j = 1; j < cant_m; j *= 2 ){
            if( threadIdx.x < cant_m / (j * 2)){
                s_vec[threadIdx.x] += s_vec[threadIdx.x + (cant_m / (j * 2))];
            }
            __syncthreads();
        }


        if ((threadIdx.x) == 0){
            detM[(offset_m / blockDim.x) + blockIdx.x] = s_vec[0];
        }
    }
}




int main(int argc, char *argv[]){


    if (argc != 3){
        printf("Falta argumento: N\n");
        printf("Falta argumento: CUDA_BLK \n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned long N = atoi (argv[1]);
    unsigned long CUDA_BLK = atoi(argv[2]),GRID_BLK,cant_blk;
    unsigned long numBytes = sizeof(BASETYPE)*4*4;
    BASETYPE *matrices,*d_matrices,*d_detM,*detM;
	double timetick;
    unsigned long i,j;
    int datos_matDet,datos_vecMult,matDet_desp;


    matrices = (BASETYPE *)malloc(numBytes*N);
    detM = (BASETYPE *)malloc(sizeof(BASETYPE)*N);
    for (i = 0; i < 4*4*N; i++){
        matrices[i] = 1;
    }

    for (i = 0; i < N; i++){
        detM[i] = 0;
    }
    matrices[2] = 220;
    matrices[13] = 220;
    matrices[7] = 6;
    matrices[14] = 6;
    //comment

    hipMalloc((void **) &d_matrices, numBytes*N);
    hipMalloc((void **) &d_detM, sizeof(BASETYPE)*N);

    datos_matDet = numBytes * CUDA_BLK + sizeof(BASETYPE) * 4 * CUDA_BLK;

    datos_vecMult = numBytes * CUDA_BLK;
    matDet_desp = CUDA_BLK * 16;

	cant_blk = N / CUDA_BLK;

    dim3 dimBlock(CUDA_BLK);
    dim3 dimGrid(cant_blk);
    
    timetick = dwalltime();


    hipMemcpy(d_matrices, matrices, numBytes*N, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_detM, detM, sizeof(BASETYPE)*N, hipMemcpyHostToDevice); // CPU -> GPU
    matDet<<<dimGrid, dimBlock,datos_matDet>>>(d_matrices,d_detM,matDet_desp);
    hipDeviceSynchronize();
    for(i = N ; i > 1; i = i / CUDA_BLK){
        GRID_BLK = i / CUDA_BLK; 
        if ((i % CUDA_BLK) == 0){
            dim3 dimGrid(GRID_BLK);
            vecMult<<<dimGrid, dimBlock,datos_vecMult>>>(d_matrices);
            hipDeviceSynchronize();
        } else{
            if(GRID_BLK != 0){
                vecMult<<<dimGrid, dimBlock,datos_vecMult>>>(d_matrices);  
                hipDeviceSynchronize(); 
            }
            dim3 dimGrid2(1);
            vecMult2<<<dimGrid2, dimBlock,datos_vecMult>>>(d_matrices,(i % CUDA_BLK),GRID_BLK * CUDA_BLK,(i % CUDA_BLK));  
            hipDeviceSynchronize();
            i = i + (i % CUDA_BLK);
        }
    }

    for(i = N ; i > 1; i = i / CUDA_BLK){
        GRID_BLK = i / CUDA_BLK; 
        if ((i % CUDA_BLK) == 0){
            dim3 dimGrid(GRID_BLK);
            sumDet<<<dimGrid, dimBlock,sizeof(BASETYPE) * 4 * CUDA_BLK>>>(d_detM);
            hipDeviceSynchronize();
        } else{
            if(GRID_BLK != 0){
                sumDet<<<dimGrid, dimBlock,sizeof(BASETYPE) * 4 * CUDA_BLK>>>(d_detM); 
                hipDeviceSynchronize(); 
            }
            dim3 dimGrid2(1);
            sumDet2<<<dimGrid, dimBlock,sizeof(BASETYPE) * 4 * CUDA_BLK>>>(d_detM,(i % CUDA_BLK),GRID_BLK * CUDA_BLK,(i % CUDA_BLK));
            hipDeviceSynchronize();
            i = i + (i % CUDA_BLK);
        }
    }

    hipDeviceSynchronize();


    hipMemcpy(matrices, d_matrices, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    hipMemcpy(detM, d_detM, sizeof(BASETYPE), hipMemcpyDeviceToHost); // GPU -> CPU

    detM[0] = detM[0] / N;

    for (i = 0; i < 16; i++){
        matrices[i] *= detM[0];
    }

	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n",error);
    
    printf("%.2lf|\n",detM[0]);

    for(i=0; i < 4; i++){
        for(j=0; j < 4; j++){
            printf("%.2lf|",matrices[i*4+j]);
        }
        printf("\n");
    }


    hipFree(d_matrices);
    hipFree(d_detM);
    free(matrices);
    free(detM);
    return 0;

}
