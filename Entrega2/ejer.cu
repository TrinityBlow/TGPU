#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

#define BASETYPE float

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

__global__ void matDet(BASETYPE *d_matA, BASETYPE *detM, int desp){ 
	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ BASETYPE datos[];
    BASETYPE *s_mat = &datos[0];
    BASETYPE *s_detAux = &datos[desp];
    int offset = (threadIdx.x)*16; 
    unsigned int i;

    for(i = 0; i < 16; i++){    
        s_mat[(threadIdx.x) * 16 + i]=d_matA[global_id * 16 + i];

    }  
    __syncthreads();  

    for(i = 0; i < 4; i++){
        s_detAux[(threadIdx.x) * 4+i]=0;
    }
    __syncthreads();

    //  printf("globalId:%d|%d|%d|%d|%d\n",global_id,(threadIdx.x)*4,(threadIdx.x)*4+1,(threadIdx.x)*4+2,(threadIdx.x)*4+3);
    s_detAux[(threadIdx.x)*4] +=  s_mat[offset] * ( (s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+13])+(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+13]))   + (-1*(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+15])) );

    s_detAux[(threadIdx.x)*4+1] +=  (-1*s_mat[offset+1]) * ( (s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+15])) );    

    s_detAux[(threadIdx.x)*4+2] +=  s_mat[offset+2] * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+15])+(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+15])) );        

    s_detAux[(threadIdx.x)*4+3] +=  (-1*s_mat[offset+3]) * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+14])+(s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+12])+(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+14])) );        
    detM[blockIdx.x * blockDim.x + (threadIdx.x)] = s_detAux[(threadIdx.x)*4] + s_detAux[(threadIdx.x)*4+1] + s_detAux[(threadIdx.x)*4+2] + s_detAux[(threadIdx.x)*4+3]; 
    __syncthreads();
        
}

__global__ void vecMult(BASETYPE *d_matA,unsigned long n){     
	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ BASETYPE s_mat[];
    unsigned int j;

	s_mat[threadIdx.x * 16]=d_matA[global_id * 16];
	s_mat[threadIdx.x * 16 + 1]=d_matA[global_id * 16 + 1];
	s_mat[threadIdx.x * 16 + 2]=d_matA[global_id * 16 + 2];
	s_mat[threadIdx.x * 16 + 3]=d_matA[global_id * 16 + 3];
	s_mat[threadIdx.x * 16 + 4]=d_matA[global_id * 16 + 4];
	s_mat[threadIdx.x * 16 + 5]=d_matA[global_id * 16 + 5];
	s_mat[threadIdx.x * 16 + 6]=d_matA[global_id * 16 + 6];
	s_mat[threadIdx.x * 16 + 7]=d_matA[global_id * 16 + 7];
	s_mat[threadIdx.x * 16 + 8]=d_matA[global_id * 16 + 8];
	s_mat[threadIdx.x * 16 + 9]=d_matA[global_id * 16 + 9];
	s_mat[threadIdx.x * 16 + 10]=d_matA[global_id * 16 + 10];
	s_mat[threadIdx.x * 16 + 11]=d_matA[global_id * 16 + 11];
	s_mat[threadIdx.x * 16 + 12]=d_matA[global_id * 16 + 12];
	s_mat[threadIdx.x * 16 + 13]=d_matA[global_id * 16 + 13];
	s_mat[threadIdx.x * 16 + 14]=d_matA[global_id * 16 + 14];
	s_mat[threadIdx.x * 16 + 15]=d_matA[global_id * 16 + 15];
	__syncthreads();

	for( j = 1; j < blockDim.x; j *= 2 ){
	    if( threadIdx.x < blockDim.x / (j * 2)){
            s_mat[(threadIdx.x) * 16] += s_mat[((threadIdx.x) * 16) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 1] += s_mat[((threadIdx.x) * 16 + 1) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 2] += s_mat[((threadIdx.x) * 16 + 2) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 3] += s_mat[((threadIdx.x) * 16 + 3) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 4] += s_mat[((threadIdx.x) * 16 + 4) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 5] += s_mat[((threadIdx.x) * 16 + 5) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 6] += s_mat[((threadIdx.x) * 16 + 6) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 7] += s_mat[((threadIdx.x) * 16 + 7) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 8] += s_mat[((threadIdx.x) * 16 + 8) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 9] += s_mat[((threadIdx.x) * 16 + 9) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 10] += s_mat[((threadIdx.x) * 16 + 10) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 11] += s_mat[((threadIdx.x) * 16 + 11) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 12] += s_mat[((threadIdx.x) * 16 + 12) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 13] += s_mat[((threadIdx.x) * 16 + 13) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 14] += s_mat[((threadIdx.x) * 16 + 14) + (blockDim.x / (j * 2)) * 16];
            s_mat[(threadIdx.x) * 16 + 15] += s_mat[((threadIdx.x) * 16 + 15) + (blockDim.x / (j * 2)) * 16];
	        
	    }
	    __syncthreads();
	}


	if ((threadIdx.x) == 0){
        d_matA[(blockIdx.x * 16)] = s_mat[0];
        d_matA[(blockIdx.x * 16) + 1] = s_mat[1];
        d_matA[(blockIdx.x * 16) + 2] = s_mat[2];
        d_matA[(blockIdx.x * 16) + 3] = s_mat[3];
        d_matA[(blockIdx.x * 16) + 4] = s_mat[4];
        d_matA[(blockIdx.x * 16) + 5] = s_mat[5];
        d_matA[(blockIdx.x * 16) + 6] = s_mat[6];
        d_matA[(blockIdx.x * 16) + 7] = s_mat[7];
        d_matA[(blockIdx.x * 16) + 8] = s_mat[8];
        d_matA[(blockIdx.x * 16) + 9] = s_mat[9];
        d_matA[(blockIdx.x * 16) + 10] = s_mat[10];
        d_matA[(blockIdx.x * 16) + 11] = s_mat[11];
        d_matA[(blockIdx.x * 16) + 12] = s_mat[12];
        d_matA[(blockIdx.x * 16) + 13] = s_mat[13];
        d_matA[(blockIdx.x * 16) + 14] = s_mat[14];
        d_matA[(blockIdx.x * 16) + 15] = s_mat[15];
	}
}

__global__ void vecMult2(BASETYPE *d_matA,unsigned long n,int offset_m,int cant_m ){     
	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ BASETYPE s_mat[];
    unsigned int j;

    if( global_id < n){

        s_mat[threadIdx.x * 16]=d_matA[(offset_m * 16) +( global_id * 16) ];
        s_mat[threadIdx.x * 16 + 1]=d_matA[(offset_m * 16) +( global_id * 16 + 1) ];
        s_mat[threadIdx.x * 16 + 2]=d_matA[(offset_m * 16) +( global_id * 16 + 2) ];
        s_mat[threadIdx.x * 16 + 3]=d_matA[(offset_m * 16) +( global_id * 16 + 3) ];
        s_mat[threadIdx.x * 16 + 4]=d_matA[(offset_m * 16) +( global_id * 16 + 4) ];
        s_mat[threadIdx.x * 16 + 5]=d_matA[(offset_m * 16) +( global_id * 16 + 5) ];
        s_mat[threadIdx.x * 16 + 6]=d_matA[(offset_m * 16) +( global_id * 16 + 6) ];
        s_mat[threadIdx.x * 16 + 7]=d_matA[(offset_m * 16) +( global_id * 16 + 7) ];
        s_mat[threadIdx.x * 16 + 8]=d_matA[(offset_m * 16) +( global_id * 16 + 8) ];
        s_mat[threadIdx.x * 16 + 9]=d_matA[(offset_m * 16) +( global_id * 16 + 9) ];
        s_mat[threadIdx.x * 16 + 10]=d_matA[(offset_m * 16) +( global_id * 16 + 10) ];
        s_mat[threadIdx.x * 16 + 11]=d_matA[(offset_m * 16) +( global_id * 16 + 11) ];
        s_mat[threadIdx.x * 16 + 12]=d_matA[(offset_m * 16) +( global_id * 16 + 12) ];
        s_mat[threadIdx.x * 16 + 13]=d_matA[(offset_m * 16) +( global_id * 16 + 13) ];
        s_mat[threadIdx.x * 16 + 14]=d_matA[(offset_m * 16) +( global_id * 16 + 14) ];
        s_mat[threadIdx.x * 16 + 15]=d_matA[(offset_m * 16) +( global_id * 16 + 15) ];
        __syncthreads();

        for( j = 1; j < cant_m; j *= 2 ){
            if( threadIdx.x < cant_m / (j * 2)){
                s_mat[(threadIdx.x) * 16] += s_mat[((threadIdx.x) * 16) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 1] += s_mat[((threadIdx.x) * 16 + 1) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 2] += s_mat[((threadIdx.x) * 16 + 2) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 3] += s_mat[((threadIdx.x) * 16 + 3) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 4] += s_mat[((threadIdx.x) * 16 + 4) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 5] += s_mat[((threadIdx.x) * 16 + 5) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 6] += s_mat[((threadIdx.x) * 16 + 6) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 7] += s_mat[((threadIdx.x) * 16 + 7) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 8] += s_mat[((threadIdx.x) * 16 + 8) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 9] += s_mat[((threadIdx.x) * 16 + 9) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 10] += s_mat[((threadIdx.x) * 16 + 10) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 11] += s_mat[((threadIdx.x) * 16 + 11) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 12] += s_mat[((threadIdx.x) * 16 + 12) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 13] += s_mat[((threadIdx.x) * 16 + 13) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 14] += s_mat[((threadIdx.x) * 16 + 14) + (cant_m / (j * 2)) * 16];
                s_mat[(threadIdx.x) * 16 + 15] += s_mat[((threadIdx.x) * 16 + 15) + (cant_m / (j * 2)) * 16];
                
            }
            __syncthreads();
        }


        if ((threadIdx.x) == 0){
            d_matA[(offset_m / blockDim.x) * 16 + (blockIdx.x * 16)] = s_mat[0];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 1] = s_mat[1];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 2] = s_mat[2];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 3] = s_mat[3];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 4] = s_mat[4];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 5] = s_mat[5];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 6] = s_mat[6];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 7] = s_mat[7];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 8] = s_mat[8];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 9] = s_mat[9];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 10] = s_mat[10];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 11] = s_mat[11];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 12] = s_mat[12];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 13] = s_mat[13];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 14] = s_mat[14];
            d_matA[((offset_m / blockDim.x) * 16 + (blockIdx.x * 16)) + 15] = s_mat[15];
        }
    }
}

int main(int argc, char *argv[]){


    if (argc != 3){
        printf("Falta argumento: N\n");
        printf("Falta argumento: CUDA_BLK \n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned long N = atoi (argv[1]);
    unsigned long CUDA_BLK = atoi(argv[2]),GRID_BLK,cant_blk;
    unsigned long numBytes = sizeof(BASETYPE)*4*4;
    BASETYPE *matrices,*d_matrices,*d_detM,*detM;
	double timetick;
    unsigned long i,j;
    int datos_matDet,datos_vecMult,matDet_desp;


    matrices = (BASETYPE *)malloc(numBytes*N);
    detM = (BASETYPE *)malloc(sizeof(BASETYPE)*N);
    for (i = 0; i < 4*4*N; i++){
        matrices[i] = 1;
    }

    for (i = 0; i < N; i++){
        detM[i] = 0;
    }
    matrices[2] = 220;
    matrices[13] = 220;
    matrices[7] = 6;
    matrices[14] = 6;
    //comment

    hipMalloc((void **) &d_matrices, numBytes*N);
    hipMalloc((void **) &d_detM, sizeof(BASETYPE)*N);

    datos_matDet = numBytes * CUDA_BLK + sizeof(BASETYPE) * 4 * CUDA_BLK;

    datos_vecMult = numBytes * CUDA_BLK;
    matDet_desp = CUDA_BLK * 16;

	cant_blk = N / CUDA_BLK;

    dim3 dimBlock(CUDA_BLK);
    dim3 dimGrid(cant_blk);
    
    timetick = dwalltime();


    hipMemcpy(d_matrices, matrices, numBytes*N, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_detM, detM, sizeof(BASETYPE)*N, hipMemcpyHostToDevice); // CPU -> GPU
    matDet<<<dimGrid, dimBlock,datos_matDet>>>(d_matrices,d_detM,matDet_desp);
    hipDeviceSynchronize();
    for(i = N ; i > 1; i = i / CUDA_BLK){
        GRID_BLK = i / CUDA_BLK; 
        if ((i % CUDA_BLK) == 0){
      //      printf("primero---------------------------------\n");
            dim3 dimGrid(GRID_BLK);
            vecMult<<<dimGrid, dimBlock,datos_vecMult>>>(d_matrices,i);
            hipDeviceSynchronize();
        } else{
            if(GRID_BLK != 0){
                vecMult<<<dimGrid, dimBlock,datos_vecMult>>>(d_matrices,i);  
                hipDeviceSynchronize(); 
            }
       //     printf("segundo---------------------------------\n");
            dim3 dimGrid2(1);
            vecMult2<<<dimGrid2, dimBlock,datos_vecMult>>>(d_matrices,(i % CUDA_BLK),GRID_BLK * CUDA_BLK,(i % CUDA_BLK));  
            hipDeviceSynchronize();
            i = i + (i % CUDA_BLK);
        }
    }
 /*   for(i = N ; i > 1; i = i / CUDA_BLK){
        GRID_BLK = i / CUDA_BLK; 
        dim3 dimGrid(GRID_BLK);
        vecMult<<<dimGrid, dimBlock,datos_vecMult>>>(d_matrices,i);
        hipDeviceSynchronize();
    }*/
    hipMemcpy(matrices, d_matrices, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    hipMemcpy(detM, d_detM, sizeof(BASETYPE)*N, hipMemcpyDeviceToHost); // GPU -> CPU

    for(i = 1; i < N ; i++){
        detM[0] += detM[i]; 
    }
    detM[0] = detM[0] / N;

    for (i = 0; i < 4*4; i++){
        matrices[i] *= detM[0];
    }

	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n",error);
    
    printf("%.2lf|\n",detM[0]);

    for(i=0; i < 4; i++){
        for(j=0; j < 4; j++){
            printf("%.2lf|",matrices[i*4+j]);
        }
        printf("\n");
    }


    hipFree(d_matrices);
    hipFree(d_detM);
    free(matrices);
    free(detM);
    return 0;

}
