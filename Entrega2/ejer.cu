#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}
__global__ void matDet(double *d_matA, double *detM, int desp){ 
	int global_id =  blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    extern __shared__ double datos[];
    double *s_mat = &datos[0];
    double *s_detAux = &datos[desp];
    int offset = (threadIdx.y * blockDim.x + threadIdx.x)*16; 
    unsigned int i;

    for(i = 0; i < 16; i++){
        s_mat[(threadIdx.y * blockDim.x + threadIdx.x) * 16 + i]=d_matA[global_id * 16 + i];

    }  
    __syncthreads();  

    for(i = 0; i < 4; i++){
        s_detAux[(threadIdx.y * blockDim.x + threadIdx.x) * 4+i]=0;
    }
    __syncthreads();

    //  printf("globalId:%d|%d|%d|%d|%d\n",global_id,(threadIdx.y * blockDim.x + threadIdx.x)*4,(threadIdx.y * blockDim.x + threadIdx.x)*4+1,(threadIdx.y * blockDim.x + threadIdx.x)*4+2,(threadIdx.y * blockDim.x + threadIdx.x)*4+3);
    s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4] +=  s_mat[offset] * ( (s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+13])+(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+13]))   + (-1*(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+15])) );

    s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+1] +=  (-1*s_mat[offset+1]) * ( (s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+15])) );    

    s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+2] +=  s_mat[offset+2] * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+15])+(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+15])) );        

    s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+3] +=  (-1*s_mat[offset+3]) * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+14])+(s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+12])+(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+14])) );        
    
    detM[blockIdx.x * blockDim.x * blockDim.y + (threadIdx.y * blockDim.x + threadIdx.x)] = s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+1] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+2] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+3]; 
    __syncthreads();
        
}

__global__ void vecMult(double *d_matA,unsigned long n, int iteraciones){      
	int global_id =  blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    extern __shared__ double s_mat[];
    unsigned int i,j;


    for(i = 0; i < 16; i++){
        s_mat[(threadIdx.y * blockDim.x + threadIdx.x) * 16 + i]=d_matA[global_id * 16 + i];
    }        
    __syncthreads();

    for( j = 1; j < (blockDim.x * blockDim.y); j *= 2 ){
        if ((threadIdx.y * blockDim.x + threadIdx.x) < blockDim.x * blockDim.y / j){

            if((threadIdx.y * blockDim.x + threadIdx.x) < blockDim.x * blockDim.y / (j * 2)){
                for( i = 0; i < 16;  i++) {
                    s_mat[(threadIdx.y * blockDim.x + threadIdx.x) * 16 + i] += s_mat[((threadIdx.y * blockDim.x + threadIdx.x) * 16 + i) + (blockDim.x * blockDim.y / (j * 2)) * 16]; // 2 * 16 = 32
                }
                if ( (threadIdx.y * blockDim.x + threadIdx.x) == 0){
                }
            }
            __syncthreads();
        }
    }


    if ((threadIdx.y * blockDim.x + threadIdx.x) == 0){
        for (i = 0; i < 16; i++){
            d_matA[(blockIdx.x * 16) + i] = s_mat[i];
        }
    }
}

int main(int argc, char *argv[]){


    if (argc != 3){
        printf("Falta argumento: N\n");
        printf("Falta argumento: CUDA_BLK \n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned long N = atoi (argv[1]);
    unsigned long CUDA_BLK = atoi(argv[2]),GRID_BLK,CUDA_BLK_2D = CUDA_BLK * CUDA_BLK;
    unsigned long numBytes = sizeof(double)*4*4;
    double *matrices,*d_matrices,*d_detM,*detM,timetick;
    unsigned long i,j;
    int iteraciones,datos_matDet,datos_vecMult,matDet_desp;


    matrices = (double *)malloc(numBytes*N);
    detM = (double *)malloc(sizeof(double)*N);
    for (i = 0; i < 4*4*N; i++){
        matrices[i] = 1;
    }

    for (i = 0; i < N; i++){
        detM[i] = 0;
    }
    matrices[2] = 220;
    matrices[13] = 220;
    matrices[7] = 6;
    matrices[14] = 6;
    //comment

    hipMalloc((void **) &d_matrices, numBytes*N);
    hipMalloc((void **) &d_detM, sizeof(double)*N);
    datos_matDet = numBytes * CUDA_BLK_2D + sizeof(double) * 4 * CUDA_BLK_2D;
    datos_vecMult = numBytes * CUDA_BLK_2D;
    matDet_desp = CUDA_BLK_2D * 16;

    dim3 dimBlock(CUDA_BLK,CUDA_BLK);
    dim3 dimGrid(N/ (CUDA_BLK_2D));
    
    timetick = dwalltime();

    iteraciones = log(CUDA_BLK_2D) / log(2);

    hipMemcpy(d_matrices, matrices, numBytes*N, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_detM, detM, sizeof(double)*N, hipMemcpyHostToDevice); // CPU -> GPU
    matDet<<<dimGrid, dimBlock,datos_matDet>>>(d_matrices,d_detM,matDet_desp);
    hipDeviceSynchronize();
    for(i = CUDA_BLK_2D ; i <= N; i *= CUDA_BLK_2D){
        GRID_BLK = N / (i / (CUDA_BLK_2D)) / (CUDA_BLK_2D); 
        dim3 dimGrid(GRID_BLK);
        printf("%d|||\n",(int)GRID_BLK);
   //     vecMult<<<dimGrid, dimBlock,datos_vecMult>>>(d_matrices,(4*4*N) / (i / CUDA_BLK_2D),iteraciones);
     //   hipDeviceSynchronize();
    }
    hipMemcpy(matrices, d_matrices, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    hipMemcpy(detM, d_detM, sizeof(double)*N, hipMemcpyDeviceToHost); // GPU -> CPU

    for(i = 1; i < N ; i++){
        detM[0] += detM[i]; 
    }
    detM[0] = detM[0] / N;

    for (i = 0; i < 4*4; i++){
        matrices[i] *= detM[0];
    }

	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n",error);
    
    printf("%.2lf|\n",detM[0]);

    for(i=0; i < 4; i++){
        for(j=0; j < 4; j++){
            printf("%.2lf|",matrices[i*4+j]);
        }
        printf("\n");
    }


    hipFree(d_matrices);
    hipFree(d_detM);
    free(matrices);
    free(detM);
    return 0;

}