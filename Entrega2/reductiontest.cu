#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

#define BASETYPE float

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

__global__ void matDet(BASETYPE *d_matA, BASETYPE *detM, int desp){ 
	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ BASETYPE datos[];
    BASETYPE *s_mat = &datos[0];
    BASETYPE *s_detAux = &datos[desp];
    int offset = (threadIdx.x)*16; 
    unsigned int i;

    for(i = 0; i < 16; i++){
        s_mat[(threadIdx.x) * 16 + i]=d_matA[global_id * 16 + i];

    }  
    __syncthreads();  

    for(i = 0; i < 4; i++){
        s_detAux[(threadIdx.x) * 4+i]=0;
    }
    __syncthreads();

    //  printf("globalId:%d|%d|%d|%d|%d\n",global_id,(threadIdx.x)*4,(threadIdx.x)*4+1,(threadIdx.x)*4+2,(threadIdx.x)*4+3);
    s_detAux[(threadIdx.x)*4] +=  s_mat[offset] * ( (s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+13])+(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+13]))   + (-1*(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+15])) );

    s_detAux[(threadIdx.x)*4+1] +=  (-1*s_mat[offset+1]) * ( (s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+15])) );    

    s_detAux[(threadIdx.x)*4+2] +=  s_mat[offset+2] * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+15])+(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+15])) );        

    s_detAux[(threadIdx.x)*4+3] +=  (-1*s_mat[offset+3]) * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+14])+(s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+12])+(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+14])) );        
    detM[blockIdx.x * blockDim.x + (threadIdx.x)] = s_detAux[(threadIdx.x)*4] + s_detAux[(threadIdx.x)*4+1] + s_detAux[(threadIdx.x)*4+2] + s_detAux[(threadIdx.x)*4+3]; 
    __syncthreads();
        
}

__global__ void vecMult(BASETYPE *d_matA,unsigned long n, int iteraciones){     
	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ BASETYPE s_mat[];
    unsigned int i,j;


    for(i = 0; i < 16; i++){
        s_mat[threadIdx.x * 16 + i]=d_matA[global_id * 16 + i];
    }
    __syncthreads();

    for( j = 1; j < blockDim.x; j *= 2 ){
        if( threadIdx.x < blockDim.x / (j * 2)){
            for( i = 0; i < 16;  i++) {
                s_mat[(threadIdx.x) * 16 + i] += s_mat[((threadIdx.x) * 16 + i) + (blockDim.x / (j * 2)) * 16]; // 2 * 16 = 32
            }
        }
        __syncthreads();
    }


    if ((threadIdx.x) == 0){
        for (i = 0; i < 16; i++){
            d_matA[(blockIdx.x * 16) + i] = s_mat[i];
        }
    }
}

int main(int argc, char *argv[]){


    if (argc != 3){
        printf("Falta argumento: N\n");
        printf("Falta argumento: CUDA_BLK \n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned long N = atoi (argv[1]);
    unsigned long CUDA_BLK = atoi(argv[2]),GRID_BLK,cant_blk;
    unsigned long numBytes = sizeof(BASETYPE)*4*4;
    BASETYPE *matrices,*d_matrices,*d_detM,*detM;
	double timetick;
    unsigned long i,j;
    int iteraciones,datos_matDet,datos_vecMult,matDet_desp;


    matrices = (BASETYPE *)malloc(numBytes*N);
    detM = (BASETYPE *)malloc(sizeof(BASETYPE)*N);
    for (i = 0; i < 4*4*N; i++){
        matrices[i] = 1;
    }

    for (i = 0; i < N; i++){
        detM[i] = 0;
    }
    matrices[2] = 220;
    matrices[13] = 220;
    matrices[7] = 6;
    matrices[14] = 6;
    //comment

    hipMalloc((void **) &d_matrices, numBytes*N);
    hipMalloc((void **) &d_detM, sizeof(BASETYPE)*N);

    datos_matDet = numBytes * CUDA_BLK + sizeof(BASETYPE) * 4 * CUDA_BLK;

    datos_vecMult = numBytes * CUDA_BLK;
    matDet_desp = CUDA_BLK * 16;

	cant_blk = N / CUDA_BLK;

    dim3 dimBlock(CUDA_BLK);
    dim3 dimGrid(cant_blk);
    
    timetick = dwalltime();

    iteraciones = log(CUDA_BLK) / log(2);

    hipMemcpy(d_matrices, matrices, numBytes*N, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_detM, detM, sizeof(BASETYPE)*N, hipMemcpyHostToDevice); // CPU -> GPU
    matDet<<<dimGrid, dimBlock,datos_matDet>>>(d_matrices,d_detM,matDet_desp);
    hipDeviceSynchronize();
    for(i = CUDA_BLK ; i <= N; i *= CUDA_BLK){
        GRID_BLK = N / (i / (CUDA_BLK)) / (CUDA_BLK); 
        dim3 dimGrid(GRID_BLK);
        vecMult<<<dimGrid, dimBlock,datos_vecMult>>>(d_matrices,(4*4*N) / (i / CUDA_BLK),iteraciones);
        hipDeviceSynchronize();
    }
    hipMemcpy(matrices, d_matrices, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    hipMemcpy(detM, d_detM, sizeof(BASETYPE)*N, hipMemcpyDeviceToHost); // GPU -> CPU

    for(i = 1; i < N ; i++){
        detM[0] += detM[i]; 
    }
    detM[0] = detM[0] / N;

    for (i = 0; i < 4*4; i++){
        matrices[i] *= detM[0];
    }

	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n",error);
    
    printf("%.2lf|\n",detM[0]);

    for(i=0; i < 4; i++){
        for(j=0; j < 4; j++){
            printf("%.2lf|",matrices[i*4+j]);
        }
        printf("\n");
    }


    hipFree(d_matrices);
    hipFree(d_detM);
    free(matrices);
    free(detM);
    return 0;

}
