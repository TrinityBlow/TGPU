#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}
__global__ void matDet(double *d_matA, double *detM){ 
	int global_id =  blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    __shared__ double s_mat[16*64];
    __shared__ double s_detAux[4*64];
    int offset = (threadIdx.y * blockDim.x + threadIdx.x)*16; 
    unsigned int i,j;

    for(i = 0; i < 16; i++){
        s_mat[(threadIdx.y * blockDim.x + threadIdx.x) * 16 + i]=d_matA[global_id * 16 + i];

    }  
    __syncthreads();  

    for(i = 0; i < 4; i++){
        s_detAux[(threadIdx.y * blockDim.x + threadIdx.x) * 4+i]=0;
    }
    __syncthreads();

    if ((threadIdx.y * blockDim.x + threadIdx.x) < 64){
        //  printf("globalId:%d|%d|%d|%d|%d\n",global_id,(threadIdx.y * blockDim.x + threadIdx.x)*4,(threadIdx.y * blockDim.x + threadIdx.x)*4+1,(threadIdx.y * blockDim.x + threadIdx.x)*4+2,(threadIdx.y * blockDim.x + threadIdx.x)*4+3);
        s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4] +=  s_mat[offset] * ( (s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+13])+(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+13]))   + (-1*(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+15])) );
    
        s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+1] +=  (-1*s_mat[offset+1]) * ( (s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+15])) );    

        s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+2] +=  s_mat[offset+2] * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+15])+(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+15])) );        

        s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+3] +=  (-1*s_mat[offset+3]) * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+14])+(s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+12])+(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+14])) );        
     
        detM[blockIdx.x*64 + (threadIdx.y * blockDim.x + threadIdx.x)] = s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+1] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+2] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+3]; 
        __syncthreads();
          
    }
}

__global__ void vecMult(double *d_matA,unsigned long n, int iteraciones){      
	int global_id =  blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    __shared__ double s_mat[16*64];
    unsigned int i,j;


    for(i = 0; i < 16; i++){
        s_mat[(threadIdx.y * blockDim.x + threadIdx.x) * 16 + i]=d_matA[global_id * 16 + i];
    }        
    __syncthreads();

    for( j = 1; j < (blockDim.x * blockDim.y); j *= 2 ){
        if ((threadIdx.y * blockDim.x + threadIdx.x) < blockDim.x * blockDim.y / j){

            if((threadIdx.y * blockDim.x + threadIdx.x) < blockDim.x * blockDim.y / (j * 2)){
                for( i = 0; i < 16;  i++) {
                    s_mat[(threadIdx.y * blockDim.x + threadIdx.x) * 16 + i] += s_mat[((threadIdx.y * blockDim.x + threadIdx.x) * 16 + i) + (blockDim.x * blockDim.y / (j * 2)) * 16]; // 2 * 16 = 32
                }
                if ( (threadIdx.y * blockDim.x + threadIdx.x) == 0){
                }
            }
            __syncthreads();
        }
    }


    if ((threadIdx.y * blockDim.x + threadIdx.x) == 0){
        for (i = 0; i < 16; i++){
            d_matA[(blockIdx.x * 16) + i] = s_mat[i];
        }
    }
}

int main(int argc, char *argv[]){


    if (argc != 2){
        printf("Falta argumento: N\n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned long N = atoi (argv[1]);
    unsigned long CUDA_BLK = 8,GRID_BLK;
    unsigned long numBytes = sizeof(double)*4*4;
    double *matrices,*d_matrices,*d_detM,*detM,timetick;
    unsigned long i,j;
    int iteraciones;


    matrices = (double *)malloc(numBytes*N);
    detM = (double *)malloc(sizeof(double)*N);
    for (i = 0; i < 4*4*N; i++){
        matrices[i] = 1;
    }

    for (i = 0; i < N; i++){
        detM[i] = 0;
    }
    matrices[2] = 220;
    matrices[13] = 220;
    matrices[7] = 6;
    matrices[14] = 6;
    //comment

    hipMalloc((void **) &d_matrices, numBytes*N);
    hipMalloc((void **) &d_detM, sizeof(double)*N);

    dim3 dimBlock(CUDA_BLK,CUDA_BLK);
    dim3 dimGrid(N/ (CUDA_BLK * CUDA_BLK));
    
    timetick = dwalltime();

    iteraciones = log(CUDA_BLK * CUDA_BLK) / log(2);

    hipMemcpy(d_matrices, matrices, numBytes*N, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_detM, detM, sizeof(double)*N, hipMemcpyHostToDevice); // CPU -> GPU
    matDet<<<dimGrid, dimBlock>>>(d_matrices,d_detM);
    hipDeviceSynchronize();
    for(i = 64 ; i <= N; i *= 64){
        GRID_BLK = N / (i / 64) / 64; 
        dim3 dimGrid(GRID_BLK);
        vecMult<<<dimGrid, dimBlock>>>(d_matrices,(4*4*N) / (i / 64),iteraciones);
        hipDeviceSynchronize();
    }
    hipMemcpy(matrices, d_matrices, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    hipMemcpy(detM, d_detM, sizeof(double)*N, hipMemcpyDeviceToHost); // GPU -> CPU

    for(i = 1; i < N ; i++){
        detM[0] += detM[i]; 
    }
    detM[0] = detM[0] / N;

    for (i = 0; i < 4*4; i++){
        matrices[i] *= detM[0];
    }

	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n",error);
    
    printf("%.2lf|\n",detM[0]);

    for(i=0; i < 4; i++){
        for(j=0; j < 4; j++){
            printf("%.2lf|",matrices[i*4+j]);
        }
        printf("\n");
    }


    hipFree(d_matrices);
    hipFree(d_detM);
    free(matrices);
    free(detM);
    return 0;

}