#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}
__global__ void matDet(double *d_matA, double *detM){ 
    
	int global_id =  blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    __shared__ double s_mat[sizeof(double)*64];
    __shared__ double s_detAux[16];
    int offset = (threadIdx.y * blockDim.x + threadIdx.x)*16; 

    if ((threadIdx.y * blockDim.x + threadIdx.x) < 64){
        s_mat[(threadIdx.y * blockDim.x + threadIdx.x)]=d_matA[global_id];
        if(threadIdx.y * blockDim.x + threadIdx.x < 16){
            s_detAux[threadIdx.y * blockDim.x + threadIdx.x] = 0;
        }
        __syncthreads();
 
        if(threadIdx.y * blockDim.x + threadIdx.x < 4){
          //  printf("globalId:%d|%d|%d|%d|%d\n",global_id,(threadIdx.y * blockDim.x + threadIdx.x)*4,(threadIdx.y * blockDim.x + threadIdx.x)*4+1,(threadIdx.y * blockDim.x + threadIdx.x)*4+2,(threadIdx.y * blockDim.x + threadIdx.x)*4+3);
            s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4] +=  s_mat[offset] * ( (s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+13])+(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+13]))   + (-1*(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+15])) );
    //        __syncthreads();       
           
            s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+1] +=  (-1*s_mat[offset+1]) * ( (s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+15])+(s_mat[offset+6]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+14])   +  (-1*(s_mat[offset+7]*s_mat[offset+10]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+14]))  + (-1*(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+15])) );    
     //       __syncthreads();

            s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+2] +=  s_mat[offset+2] * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+15])+(s_mat[offset+5]*s_mat[offset+11]*s_mat[offset+12])+(s_mat[offset+7]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+7]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+11]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+15])) );        
     //       __syncthreads();
      
            s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+3] +=  (-1*s_mat[offset+3]) * ( (s_mat[offset+4]*s_mat[offset+9]*s_mat[offset+14])+(s_mat[offset+5]*s_mat[offset+10]*s_mat[offset+12])+(s_mat[offset+6]*s_mat[offset+8]*s_mat[offset+13])   +  (-1*(s_mat[offset+6]*s_mat[offset+9]*s_mat[offset+12]))   + (-1*(s_mat[offset+4]*s_mat[offset+10]*s_mat[offset+13]))  + (-1*(s_mat[offset+5]*s_mat[offset+8]*s_mat[offset+14])) );            
       //     __syncthreads();
          
            detM[blockIdx.x*4 + (threadIdx.y * blockDim.x + threadIdx.x)] = s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+1] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+2] + s_detAux[(threadIdx.y * blockDim.x + threadIdx.x)*4+3]; 
  //          __syncthreads();
          
        }
    }
}

__global__ void vecMult(double *d_matA,unsigned long n){      
	int global_id =  blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    __shared__ double s_mat[sizeof(double)*64];
    unsigned int i;

    if ((threadIdx.y * blockDim.x + threadIdx.x) < 64){
        s_mat[(threadIdx.y * blockDim.x + threadIdx.x)]=d_matA[global_id];
        __syncthreads();



        for( i = 1; i <= 2;  i++) {
            if(( threadIdx.y * blockDim.x + threadIdx.x )< (int)(64 >> i)){
                s_mat[(threadIdx.y * blockDim.x + threadIdx.x)] += s_mat[((threadIdx.y * blockDim.x + threadIdx.x ) + (64 >> i))];
            }
            __syncthreads();
        }

        if ( (threadIdx.y * blockDim.x + threadIdx.x) < 16){
            d_matA[blockIdx.x * 16 + (threadIdx.y * blockDim.x + threadIdx.x)] = s_mat[(threadIdx.y * blockDim.x + threadIdx.x)];
        }
    }
}



__global__ void vecMult3(double *d_matA,unsigned long n,int iteraciones){      
	int global_id =  blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    __shared__ double s_mat[sizeof(double)*128];
    unsigned int i;
        /*
        for( i = 1; i <= 2;  i++) {
            if(( threadIdx.y * blockDim.x + threadIdx.x )< (int)(64 >> i)){
                s_mat[(threadIdx.y * blockDim.x + threadIdx.x)] += s_mat[((threadIdx.y * blockDim.x + threadIdx.x ) + (64 >> i))];
            }
            __syncthreads();
        }
        */
    for(i = 1 ; i <= iteraciones ; i++){
        if ( global_id < ( n / (1 << (i) ))){
            s_mat[(threadIdx.y * blockDim.x + threadIdx.x)]=d_matA[(((int)(global_id / 16))*32 + (global_id % 16))];
            s_mat[(threadIdx.y * blockDim.x + threadIdx.x)+64]=d_matA[((int)(global_id / 16)*32 + (global_id % 16)+ 16)];
         //   printf("global:%d||%.2lf||%.2lf||\n",global_id,s_mat[(threadIdx.y * blockDim.x + threadIdx.x)],s_mat[(threadIdx.y * blockDim.x + threadIdx.x)+64]);
            printf("global:%d||%d||\n",global_id,(((int)(global_id / 16))*32 + (global_id % 16)));
            __syncthreads();

            
            s_mat[(threadIdx.y * blockDim.x + threadIdx.x)] += s_mat[(threadIdx.y * blockDim.x + threadIdx.x) + 64];
           __syncthreads();

            d_matA[global_id] = s_mat[(threadIdx.y * blockDim.x + threadIdx.x)];
            __syncthreads();
            if(global_id == 0){
                printf("-------------------------------------------\n");

            }
        }
    }
}

__global__ void vecMult2(double *d_matA,unsigned long n,int iteraciones){      
	int global_id =  blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    __shared__ double s_mat[128];
    unsigned int i;
        /*
        for( i = 1; i <= 2;  i++) {
            if(( threadIdx.y * blockDim.x + threadIdx.x )< (int)(64 >> i)){
                s_mat[(threadIdx.y * blockDim.x + threadIdx.x)] += s_mat[((threadIdx.y * blockDim.x + threadIdx.x ) + (64 >> i))];
            }
            __syncthreads();
        }
        */
    for(i = 1 ; i <= iteraciones ; i++){
        if ( global_id < ( n / (1 << (i) ))){
            s_mat[(threadIdx.y * blockDim.x + threadIdx.x)]=d_matA[global_id];
            __syncthreads();
            s_mat[(threadIdx.y * blockDim.x + threadIdx.x)+64]=d_matA[(int)(global_id + ( n / (1 << (i) )))];
            __syncthreads();
            printf("global:%d||%.2lf||%.2lf||\n",global_id,s_mat[(threadIdx.y * blockDim.x + threadIdx.x)],s_mat[(threadIdx.y * blockDim.x + threadIdx.x)+64]);

            __syncthreads();

         /*   if(global_id == 0){
                printf("global:%d||%d||suma:%.2lf v1:%.2lf v2:%.2lf\n",global_id,(int)(global_id + ( n / (1 << (i) ))),s_mat[(threadIdx.y * blockDim.x + threadIdx.x)]+s_mat[(threadIdx.y * blockDim.x + threadIdx.x) + 64],s_mat[(threadIdx.y * blockDim.x + threadIdx.x)],s_mat[(threadIdx.y * blockDim.x + threadIdx.x) + 64]);
            }*/
            s_mat[(threadIdx.y * blockDim.x + threadIdx.x)] += s_mat[(threadIdx.y * blockDim.x + threadIdx.x) + 64];           

           __syncthreads();

            d_matA[global_id] = s_mat[(threadIdx.y * blockDim.x + threadIdx.x)];
         __syncthreads();
        /*    if(global_id == 0){
                printf("-------------------------------------------\n");

            }*/
        }
         __syncthreads();
    }
}





int main(int argc, char *argv[]){


    if (argc != 2){
        printf("Falta argumento: N\n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned long N = atoi (argv[1]);
    unsigned long CUDA_BLK = 8,GRID_BLK;
    unsigned long numBytes = sizeof(double)*4*4;
    double *matrices,*d_matrices,*d_detM,*detM,timetick;
    unsigned long i,j;
    int iteraciones;


    matrices = (double *)malloc(numBytes*N);
    detM = (double *)malloc(sizeof(double)*N);
    for (i = 0; i < 4*4*N; i++){
        matrices[i] = 1;
    }

    for (i = 0; i < N; i++){
        detM[i] = 0;
    }
    matrices[2] = 220;
    matrices[13] = 220;
    matrices[7] = 6;
    matrices[14] = 6;
    //comment

    hipMalloc((void **) &d_matrices, numBytes*N);
    hipMalloc((void **) &d_detM, sizeof(double)*N);

    dim3 dimBlock(CUDA_BLK,CUDA_BLK);
    dim3 dimGrid(N/4);
    
    timetick = dwalltime();

    iteraciones = log(N) / log(2);

    hipMemcpy(d_matrices, matrices, numBytes*N, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_detM, detM, sizeof(double)*N, hipMemcpyHostToDevice); // CPU -> GPU
    matDet<<<dimGrid, dimBlock>>>(d_matrices,d_detM);
    hipDeviceSynchronize();
   /* for(i = 4 ; i <= N; i *= 4){
        GRID_BLK = N / i; 
        dim3 dimGrid(GRID_BLK);
        vecMult<<<dimGrid, dimBlock>>>(d_matrices,(4*4*N) / (i / 2));
        hipDeviceSynchronize();
    }*/
    dim3 dimGrid2(4*4*N/CUDA_BLK/CUDA_BLK / 2);
    vecMult2<<<dimGrid2, dimBlock>>>(d_matrices,(4*4*N),iteraciones);
    hipDeviceSynchronize();
    hipMemcpy(matrices, d_matrices, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    hipMemcpy(detM, d_detM, sizeof(double)*N, hipMemcpyDeviceToHost); // GPU -> CPU

    for(i = 1; i < N ; i++){
        detM[0] += detM[i]; 
    }
    detM[0] = detM[0] / N;

    for (i = 0; i < 4*4; i++){
        matrices[i] *= detM[0];
    }

	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n",error);
    
    printf("%.2lf|\n",detM[0]);

    for(i=0; i < 4; i++){
        for(j=0; j < 4; j++){
            printf("%.2lf|",matrices[i*4+j]);
        }
        printf("\n");
    }


    hipFree(d_matrices);
    hipFree(d_detM);
    free(matrices);
    free(detM);
    return 0;

}