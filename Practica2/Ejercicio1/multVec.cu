#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}



__global__ void vecPromedio(double *d_vecA,unsigned long dist,unsigned long n,unsigned long tam_tot){    
    unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < n){
        d_vecA[global_id*dist] = d_vecA[global_id*dist] * d_vecA[global_id*dist+dist / 2];
    }
}



int main(int argc, char *argv[]){


    if (argc != 3){
        printf("Falta argumento: N, CUDABLK\n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned int N = atoi (argv[1]);
    unsigned long CUDA_BLK = atoi (argv[2]);
    unsigned long numBytes = sizeof(double)*N,tam_tot;
    double *vecA,*d_vecA,*d_vecPromedio,*d_parcialA,resultgpu,timetick;
    unsigned int i;


    vecA = (double *)malloc(numBytes);
    for (i = 0; i < N; i++){
        vecA[i] = i;
    }

  tam_tot = N;
  hipMalloc((void **) &d_vecA, numBytes);
  hipMalloc((void **) &d_vecPromedio, numBytes);
  hipMalloc((void **) &d_parcialA, numBytes);

    for (i = 0; i < N; i++){
        vecA[i] = 2;
    }
    hipMemcpy(d_vecA, vecA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    //--------------------------------gpu comienza ------------------------------------

    dim3 dimBlock(CUDA_BLK);  
	timetick = dwalltime();
    hipMemcpy(d_vecPromedio, vecA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU


    //promedio
    for(i = 2; i <= N ;i *= 2){
        dim3 dimGrid((N / i + dimBlock.x - 1) / dimBlock.x);
        vecPromedio<<<dimGrid, dimBlock>>>(d_vecPromedio,i,N/i,tam_tot);
        hipDeviceSynchronize();
    }
    hipMemcpy(&resultgpu, d_vecPromedio, sizeof(double), hipMemcpyDeviceToHost); // GPU -> CPU

	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n\n",error);

    //--------------------------------gpu termina ------------------------------------

/*
    printf("----------------------------------------\n\n");
    for(i = 0; i < N; i++){
        printf("%f|",vecA[i]);
    }
	printf("\n\n");
    printf("parcialA: %f||parcialB: %f\n",parcialA,parcialB);*/

    printf("resultadoGPU: %f\n",resultgpu);

    hipFree(d_vecA);
    free(vecA);
    return 0;
}