#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}



__global__ void vecMult(double *d_vecA,unsigned long dist,unsigned long n){    
    unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < n){
        d_vecA[global_id*dist] = d_vecA[global_id*dist] * d_vecA[global_id*dist+dist / 2];
    }
}





int main(int argc, char *argv[]){


    if (argc != 2){
        printf("Falta argumento: N\n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned int N = atoi (argv[1]);
    unsigned long CUDA_BLK = 128;
    unsigned long numBytes = sizeof(double)*N;
    double *vecA,result,*d_vecA,timetick;
    unsigned int i;


    vecA = (double *)malloc(numBytes);
    result = 0;
    for (i = 0; i < N; i++){
        vecA[i] = 2;
    }

    hipMalloc((void **) &d_vecA, numBytes);

    // Bloque unidimencional de hilos (*cb* hilos)
    dim3 dimBlock(CUDA_BLK);
    //promedio
    timetick = dwalltime();
    hipMemcpy(d_vecA, vecA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    for(i = 2; i <= N ;i *= 2){
        dim3 dimGrid((N / i + dimBlock.x - 1) / dimBlock.x);
        vecMult<<<dimGrid, dimBlock>>>(d_vecA,i,N/i);
        hipDeviceSynchronize();
    }
    hipMemcpy(&result, d_vecA, sizeof(double), hipMemcpyDeviceToHost); // GPU -> CPU

	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n\n",error);
    printf("resultadoGPU: %f\n",result);

    hipFree(d_vecA);
    free(vecA);
    return 0;

}