#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}



__global__ void vecMult(double *d_vecA,unsigned long n){      
    unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ double s_vecA[sizeof(double)*32];
    unsigned int i;
 //   int y = 2;


    if (global_id < n){
        
        s_vecA[threadIdx.x]=d_vecA[global_id];
        __syncthreads();



        for( i = 1; i <= 7;  i++) {
            if( threadIdx.x < (int)(128 >>(i))){
                s_vecA[threadIdx.x * (1 << i)] += s_vecA[(threadIdx.x * (1 << i)) + (1 << (i-1))];
            }
 //           y = y * 2;
            __syncthreads();
        }

        if ( threadIdx.x == 0){
            d_vecA[blockIdx.x] = s_vecA[0];
        } 
    }
}





int main(int argc, char *argv[]){


    if (argc != 2){
        printf("Falta argumento: N\n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned long N = atoi (argv[1]);
    unsigned long CUDA_BLK = 128,GRID_BLK;
    unsigned long numBytes = sizeof(double)*N;
    double *vecA,result,*d_vecA,timetick;
    unsigned long i,j;


    vecA = (double *)malloc(numBytes);
    result = 1;
    for (i = 0; i < N; i++){
        vecA[i] = 2;
    }
    //comment

    hipMalloc((void **) &d_vecA, numBytes);
    hipMemcpy(d_vecA, vecA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU

    dim3 dimBlock(CUDA_BLK);
    
    unsigned long int iteraciones = log(N) / log(2);
    timetick = dwalltime();
    hipMemcpy(d_vecA, vecA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    for(i = N ; i > 1; i = ceil(float(i) / CUDA_BLK)){
        GRID_BLK = ceil(float(i) / CUDA_BLK) ; 
        dim3 dimGrid(GRID_BLK);
        vecMult<<<dimGrid, dimBlock>>>(d_vecA,i);
        hipDeviceSynchronize();
    }
    hipMemcpy(vecA, d_vecA, sizeof(double), hipMemcpyDeviceToHost); // GPU -> CPU

	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n",error);
    
    printf("%f|",vecA[0]);
    printf("\n\n");


    hipFree(d_vecA);
    free(vecA);
    return 0;

}