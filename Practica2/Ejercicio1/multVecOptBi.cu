#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}



__global__ void vecMult(double *d_vecA,unsigned long dist,unsigned long n,unsigned long tam_tot){    
    unsigned long int global_id = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    if (global_id < n){
        d_vecA[global_id*dist] = d_vecA[global_id*dist] * d_vecA[global_id*dist+dist / 2];
    }
}





int main(int argc, char *argv[]){


    if (argc != 2){
        printf("Falta argumento: N\n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned int N = atoi (argv[1]);
    unsigned long CUDA_BLK = 8,gridBlock; //8*8 = 64
    unsigned long numBytes = sizeof(double)*N,tam_tot;
    double *vecA,result,*d_vecA,timetick;
    unsigned int i;


    vecA = (double *)malloc(numBytes);
    result = 0;
    for (i = 0; i < N; i++){
        vecA[i] = 2;
    }


    tam_tot = N;
    hipMalloc((void **) &d_vecA, numBytes);

    // Bloque unidimencional de hilos (*cb* hilos)
    dim3 dimBlock(CUDA_BLK,CUDA_BLK); // Bloque bidimencional de hilos (*cb* hilos)
    //promedio
    timetick = dwalltime();
    hipMemcpy(d_vecA, vecA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    for(i = 2; i <= N ;i *= 2){
        gridBlock = (unsigned int)sqrt(N*N/CUDA_BLK/CUDA_BLK / i);
        dim3 dimGrid(gridBlock,gridBlock); // Grid bidimencional (*ceil(n/cb)* bloques)
        vecMult<<<dimGrid, dimBlock>>>(d_vecA,i,N/i,tam_tot);
        hipDeviceSynchronize();
    }
    hipMemcpy(&result, d_vecA, sizeof(double), hipMemcpyDeviceToHost); // GPU -> CPU

	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n\n",error);
    printf("resultadoGPU: %f\n",result);

    hipFree(d_vecA);
    free(vecA);
    return 0;

}