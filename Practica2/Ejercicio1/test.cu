#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}



__global__ void vecMult(double *d_vecA,unsigned long n, unsigned long blockSize){   
    __shared__ double sdata[sizeof(double)*128];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + tid;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;   
    __syncthreads();
    while (i < n) { sdata[tid] += d_vecA[i] + d_vecA[i+blockSize]; i += gridSize; }
    __syncthreads();
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) {
        if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
        if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
        if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
        if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
        if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
        if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
    }
    if (tid == 0) d_vecA[blockIdx.x] = sdata[0];

}





int main(int argc, char *argv[]){


    if (argc != 2){
        printf("Falta argumento: N\n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned long N = atoi (argv[1]);
    unsigned long CUDA_BLK = 128,GRID_BLK;
    unsigned long numBytes = sizeof(double)*N;
    double *vecA,result,*d_vecA,timetick;
    unsigned long i;


    vecA = (double *)malloc(numBytes);
    result = 1;
    for (i = 0; i < N; i++){
        vecA[i] = 2;
    }
    //comment

    hipMalloc((void **) &d_vecA, numBytes);

    // Bloque unidimencional de hilos (*cb* hilos)
    dim3 dimBlock(CUDA_BLK);
    //promedio
    timetick = dwalltime();
    hipMemcpy(d_vecA, vecA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    for(i = N ; i > 1; i /= CUDA_BLK){
        printf("%lu %lu\n\n",i,CUDA_BLK);
        GRID_BLK = i / CUDA_BLK ; 
        dim3 dimGrid(GRID_BLK);
        vecMult<<<dimGrid, dimBlock>>>(d_vecA,i,CUDA_BLK);
        hipDeviceSynchronize();
    }
    hipMemcpy(vecA, d_vecA, sizeof(double)*GRID_BLK, hipMemcpyDeviceToHost); // GPU -> CPU

  
    for (i = 0; i < GRID_BLK; i++){
        result *= vecA[i];
    }

	printf("Tiempo para la GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n\n",error);
    
/*
    for (i = 0; i < GRID_BLK; i++){
        printf("%f|",vecA[i]);
    }
    printf("\n\n");*/
    printf("%f|",result);
    printf("\n\n");


    hipFree(d_vecA);
    free(vecA);
    return 0;

}