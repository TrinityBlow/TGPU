#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

//134217728

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

__global__ void ecuacion_kernel_outplace_p1(double *d_matA,double *d_matAT,double *d_matB,double *d_matBT, unsigned int n){   
    int distA = blockIdx.y * blockDim.y + threadIdx.y; //i
    int distB = blockIdx.x * blockDim.x + threadIdx.x; //j
    
    //transpuesta out-place A y B
    if( (distA<n*n) && (distB<n*n) ){
        d_matAT [distB*n + distA] = d_matA[distA*n + distB];
        d_matBT [distB*n + distA] = d_matB[distA*n + distB];
    }

    
}

__global__ void ecuacion_kernel_outplace_p2(double *d_matA,double *d_matB,double *d_matC,double *d_matAT,double *d_matBT, unsigned int n){    
    int distA = blockIdx.y * blockDim.y + threadIdx.y; //i
    int distB = blockIdx.x * blockDim.x + threadIdx.x; //j
    int k;
    if (distA*n+distB < n*n){
        //multiplicacion 
        for(k = 0; k < n ;k++){
            d_matC[distA*n+distB] += d_matA[distA*n+k] * d_matBT[distB+k*n];
        }
        //suma
        if (distA*n+distB < n*n){
            d_matC[distA*n+distB] += d_matB[distA*n+distB] + d_matAT[distA*n+distB];
        }
    }
}
__global__ void ecuacion_kernel_inplace (double *d_matA,double *d_matB,double *d_matC, unsigned int n){
    int distA = blockIdx.y * blockDim.y + threadIdx.y; //i
    int distB = blockIdx.x * blockDim.x + threadIdx.x; //j
    int k;
    //multiplicacion 
    if (distA*n+distB < n*n){
        for(k = 0; k < n ;k++){
            d_matC[distA*n+distB] += d_matA[distA*n+k] * d_matB[distB*n+k];
        }
        if (distA*n+distB < n*n){
            d_matC[distA*n+distB] += d_matB[distA*n+distB] + d_matA[distA+distB*n];
        }
    }
}

void checkparams(unsigned long *n, unsigned int *cb);

int main(int argc, char *argv[]){

        if (argc != 2){
            printf("Falta argumento: N\n");
            return 0;
        }
    hipError_t error;

    unsigned int N = atoi (argv[1]);
    unsigned int CUDA_BLK = 16, gridBlock;
    unsigned long numBytes = sizeof(double)*N*N;
    double *matA,*matB,*matC,*matAT,*d_matA,*d_matB,*d_matC,*d_matAT,*d_matBT,timetick;
    unsigned int i,j,k;


    matA = (double *)malloc(numBytes);
    matAT = (double *)malloc(numBytes);
    matB = (double *)malloc(numBytes);
    matC = (double *)malloc(numBytes);

    for (i = 0; i < N*N; i++){
        matA[i] = i;
        matB[i] = i;
        matC[i] = 0;
        matAT[i] = 0;
    }

  hipMalloc((void **) &d_matA, numBytes);
  hipMalloc((void **) &d_matAT, numBytes);
  hipMalloc((void **) &d_matB, numBytes);
  hipMalloc((void **) &d_matBT, numBytes);
  hipMalloc((void **) &d_matC, numBytes);
  hipMemcpy(d_matA, matA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  hipMemcpy(d_matB, matB, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  hipMemcpy(d_matC, matC, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  gridBlock = (unsigned int)sqrt(N*N/CUDA_BLK/CUDA_BLK);

  // Bloque bidimencional de hilos (*cb* hilos)
  dim3 dimBlock(CUDA_BLK,CUDA_BLK);
  // Grid bidimencional (*ceil(n/cb)* bloques)
  dim3 dimGrid(gridBlock,gridBlock);

    //--------------------------------cpu comienza ------------------------------------

    //secuencial
	timetick = dwalltime();
    //transpuesta out-place A
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            matAT [j*N + i] = matA[i*N + j];
        }
    }
    //multiplicacion
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            for(k = 0; k < N ;k++){
                matC[i*N+j] += matA[i*N+k] * matB[j*N+k]; //multiplica a matB por fila, eso simula la matB transpuesta
            }
        }
    }
    //suma
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            matC[i*N+j] += matB[i*N+j] + matAT[i*N+j];
        }
    }
	printf("Tiempo para la ecuacion CPU: %f\n\n",dwalltime() - timetick);

    //--------------------------------cpu termina ------------------------------------



    for (i = 0; i < N*N; i++){
        matA[i] = i;
        matB[i] = i;
        matC[i] = 0;
        matAT[i] = 0;
    }

    //--------------------------------gpu out-place comienza ------------------------------------

	timetick = dwalltime();
    ecuacion_kernel_outplace_p1<<<dimGrid, dimBlock>>>(d_matA, d_matAT,d_matB,d_matBT, N);
    hipDeviceSynchronize();
    ecuacion_kernel_outplace_p2<<<dimGrid, dimBlock>>>(d_matA, d_matB,d_matC,d_matAT,d_matBT, N);
    hipDeviceSynchronize();
	printf("Tiempo para la ecuacion out-place GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n\n",error);
    
    //--------------------------------gpu out-place termina ------------------------------------

    hipMemcpy(d_matA, matA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_matB, matB, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_matC, matC, numBytes, hipMemcpyHostToDevice); // CPU -> GPU


    //--------------------------------gpu in-place comienza ------------------------------------

	timetick = dwalltime();
    ecuacion_kernel_inplace<<<dimGrid, dimBlock>>>(d_matA, d_matB,d_matC, N);
    hipDeviceSynchronize();
	printf("Tiempo para la ecuacion in-place GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n\n",error);

    //--------------------------------gpu in-place termina ------------------------------------

    hipMemcpy(matC, d_matC, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    hipMemcpy(matA, d_matA, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    hipMemcpy(matB, d_matB, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    hipMemcpy(matAT, d_matAT, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
  
/*
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matA[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");

    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matB[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");

    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matAT[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");


    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matC[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");
*/

    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);
    hipFree(d_matAT);
    hipFree(d_matBT);
    free(matA);
    free(matB);
    free(matC);
    free(matAT);
    return 0;
}

void checkparams(unsigned long *n, unsigned int *cb){
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %lu (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}
