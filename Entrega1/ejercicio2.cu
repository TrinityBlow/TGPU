#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}



__global__ void vecPromedio(double *d_vecA,unsigned long dist,unsigned long n,unsigned long tam_tot){    
    unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < n){
        d_vecA[global_id*dist] = d_vecA[global_id*dist] + d_vecA[global_id*dist+dist / 2];
        if(dist == tam_tot) {
            d_vecA[global_id*dist] /= tam_tot;
        }
    }
}

__global__ void acomulativo(double *d_parcialA,double *d_parcialB,unsigned long dist,unsigned long n,unsigned long tam_tot){    
    unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < n){
        d_parcialA[global_id*dist] = d_parcialA[global_id*dist] + d_parcialA[global_id*dist+dist / 2];
        d_parcialB[global_id*dist] = d_parcialB[global_id*dist] + d_parcialB[global_id*dist+dist / 2];
        if(dist == tam_tot) {
            d_parcialB[0] += 1;
            d_parcialB[0] = sqrt(d_parcialA[0] / d_parcialB[0]); 
        }
    }
}

__global__ void sumatoria(double *d_parcialA,double *d_parcialB,double *d_vecPromedio, unsigned long n){    
    unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < n){
        d_parcialA[global_id] = (d_parcialB[global_id] - d_vecPromedio[0]) * (d_parcialB[global_id] - d_vecPromedio[0]);
        d_parcialB[global_id] = (d_parcialB[global_id] + d_vecPromedio[0]) * (d_parcialB[global_id] + d_vecPromedio[0]);
    }
}




int main(int argc, char *argv[]){


    if (argc != 3){
        printf("Falta argumento: N, CUDABLK\n");
        return 0;
    }
	//declaracion de variables
    hipError_t error;
    unsigned int N = atoi (argv[1]);
    unsigned long CUDA_BLK = atoi (argv[2]);
    unsigned long numBytes = sizeof(double)*N,tam_tot;
    double *vecA,promedio,result,parcialA,parcialB,*d_vecA,*d_vecPromedio,*d_parcialA,resultgpu,timetick;
    unsigned int i;


    vecA = (double *)malloc(numBytes);
    promedio = 0;
    result = 0;
    parcialA = 0;
    parcialB = 0;
    for (i = 0; i < N; i++){
        vecA[i] = i;
    }

  hipMalloc((void **) &d_vecA, numBytes);
  hipMalloc((void **) &d_vecPromedio, numBytes);
  hipMalloc((void **) &d_parcialA, numBytes);
  hipMemcpy(d_vecA, vecA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  hipMemcpy(d_vecPromedio, vecA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  tam_tot = N;

  // Bloque unidimencional de hilos (*cb* hilos)
  dim3 dimBlock(CUDA_BLK);

    //--------------------------------cpu comienza ------------------------------------

    //secuencial
	timetick = dwalltime();
    for (i = 0; i < N; i++){
        promedio += vecA[i];
    }
    promedio /= N;

    for (i = 0; i < N; i++){
        parcialA += (vecA[i] - promedio) * (vecA[i] - promedio);
        parcialB += (vecA[i] + promedio) * (vecA[i] + promedio);
    }
    parcialB += 1;

    result = sqrt(parcialA / parcialB);
	printf("Tiempo para la ecuacion CPU: %f\n\n",dwalltime() - timetick);

    //--------------------------------cpu termina ------------------------------------

    for (i = 0; i < N; i++){
        vecA[i] = i;
    }
    //--------------------------------gpu comienza ------------------------------------

	timetick = dwalltime();

    //promedio
    for(i = 2; i <= N ;i *= 2){
        dim3 dimGrid((N / i + dimBlock.x - 1) / dimBlock.x);
        vecPromedio<<<dimGrid, dimBlock>>>(d_vecPromedio,i,N/i,tam_tot);
        hipDeviceSynchronize();
    }

    // Grid unidimencional (*ceil(n/cb)* bloques)
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);

    //sumatoria
    sumatoria<<<dimGrid, dimBlock>>>(d_parcialA,d_vecA,d_vecPromedio,N);
    hipDeviceSynchronize();

    //sumatoria acumulativo
    for(i = 2; i <= N ;i *= 2){
        dim3 dimGrid((N / i + dimBlock.x - 1) / dimBlock.x);
        acomulativo<<<dimGrid, dimBlock>>>(d_parcialA,d_vecA,i,N/i,tam_tot);
        hipDeviceSynchronize();
    }

	printf("Tiempo para la ecuacion in-place GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n\n",error);
    hipMemcpy(&resultgpu, d_vecA, sizeof(double), hipMemcpyDeviceToHost); // GPU -> CPU

    //--------------------------------gpu termina ------------------------------------

    hipMemcpy(vecA, d_vecA, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
/*
    printf("----------------------------------------\n\n");
    for(i = 0; i < N; i++){
        printf("%f|",vecA[i]);
    }
	printf("\n\n");
    printf("promedio: %f\n",promedio);
    printf("parcialA: %f||parcialB: %f\n",parcialA,parcialB);*/
    printf("resultadoCPU: %f\n",result);
    printf("resultadoGPU: %f\n",resultgpu);

    hipFree(d_vecA);
    free(vecA);
    return 0;
}