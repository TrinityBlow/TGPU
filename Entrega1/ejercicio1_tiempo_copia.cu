#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

//134217728

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

__global__ void ecuacion_kernel_outplace_p1(double *d_matA,double *d_matAT,double *d_matB,double *d_matBT, unsigned int n){   
    int distA = blockIdx.y * blockDim.y + threadIdx.y; //i
    int distB = blockIdx.x * blockDim.x + threadIdx.x; //j
    
    //transpuesta out-place A y B
    if( (distA<n*n) && (distB<n*n) ){
        d_matAT [distB*n + distA] = d_matA[distA*n + distB];
        d_matBT [distB*n + distA] = d_matB[distA*n + distB];
    }

    
}

__global__ void ecuacion_kernel_outplace_p2(double *d_matA,double *d_matB,double *d_matC,double *d_matAT,double *d_matBT, unsigned int n){    
    int distA = blockIdx.y * blockDim.y + threadIdx.y; //i
    int distB = blockIdx.x * blockDim.x + threadIdx.x; //j
    int k;
    if (distA*n+distB <= (n*n - 1)){
        //multiplicacion 
        for(k = 0; k < n ;k++){
            d_matC[distA*n+distB] += d_matA[distA*n+k] * d_matBT[distB+k*n];
        }
        //suma
        d_matC[distA*n+distB] += d_matB[distA*n+distB] + d_matAT[distA*n+distB];
    }
}

__global__ void ecuacion_kernel_inplace_suma (double *d_matA,double *d_matB,double *d_matC, unsigned int n){
    int distA = blockIdx.y * blockDim.y + threadIdx.y; //i
    int distB = blockIdx.x * blockDim.x + threadIdx.x; //j
    int k;
    //multiplicacion 
    if (distA*n+distB < (n*n - 1)){
        d_matC[distA*n+distB] += d_matB[distA*n+distB] + d_matA[distA+distB*n]; 
        for(k = 0; k < n ;k++){
            d_matC[distA*n+distB] += d_matA[distA*n+k] * d_matB[distB*n+k]; 
        }
    }

}

__global__ void kernel_sum_Matriz (double *d_matA,double *d_matB,double *d_matC, unsigned int n){
    int distA = blockIdx.y * blockDim.y + threadIdx.y; //i
    int distB = blockIdx.x * blockDim.x + threadIdx.x; //j
    //suma 
    if (distA*n+distB < (n*n)){
        d_matC[distA*n+distB] += d_matA[distA*n+distB] + d_matB[distA+distB*n]; 
    }

}

__global__ void kernel_transpuesta(double *m, int N){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int i = int((1 + sqrtf(1 + 8*tid)) / 2);
	int j = tid - (i*(i-1)/2); int aux;
	if ( (i<N) && (j<N) ){
		aux = m[i*N + j] ;
		m[i*N + j] = m[j*N + i];
		m[j*N + i] = aux;
	}
}


__global__ void kernel_mult_sum_matriz (double *d_matA,double *d_matB,double *d_matC, unsigned int n){
    int distA = blockIdx.y * blockDim.y + threadIdx.y; //i
    int distB = blockIdx.x * blockDim.x + threadIdx.x; //j
    int k;
    //multiplicacion 
    if (distA*n+distB < (n*n)){
        for(k = 0; k < n ;k++){
            d_matC[distA*n+distB] += d_matA[distA*n+k] * d_matB[distB*n+k]; 
        }
    }

}

int main(int argc, char *argv[]){

        if (argc != 3){
            printf("Falta argumento: N, CUDABLK\n");
            return 0;
        }
	//declaracion de variables
    hipError_t error;
    unsigned int N = atoi (argv[1]);
    unsigned long CUDA_BLK = atoi (argv[2]), gridBlock;
    unsigned long numBytes = sizeof(double)*N*N;
    double *matA,*matB,*matC,*d_matA,*d_matB,*d_matC,*d_matAT,*d_matBT,timetick;
    unsigned int i,j,k;

	//inicializa variables para cpu
    matA = (double *)malloc(numBytes);
    matB = (double *)malloc(numBytes);
    matC = (double *)malloc(numBytes);
    for (i = 0; i < N*N; i++){
        matA[i] = i;
        matB[i] = i;
        matC[i] = 0;
    }

  //inicializa variables para gpu
  hipMalloc((void **) &d_matA, numBytes);
  hipMalloc((void **) &d_matAT, numBytes);
  hipMalloc((void **) &d_matB, numBytes);
  hipMalloc((void **) &d_matBT, numBytes);
  hipMalloc((void **) &d_matC, numBytes);

  gridBlock = (unsigned int)sqrt(N*N/CUDA_BLK/CUDA_BLK);
  dim3 dimBlock(CUDA_BLK,CUDA_BLK); // Bloque bidimencional de hilos (*cb* hilos)
  dim3 dimGrid(gridBlock,gridBlock); // Grid bidimencional (*ceil(n/cb)* bloques)

    //--------------------------------cpu comienza ------------------------------------

    //secuencial
	timetick = dwalltime();

    //multiplicacion
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            for(k = 0; k < N ;k++){
                matC[i*N+j] += matA[i*N+k] * matB[j*N+k]; //multiplica a matB por fila, eso simula la matB transpuesta
            }
        }
    }
    //suma
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            matC[i*N+j] += matB[i*N+j] + matA[i+j*N];
        }
    }

	printf("Tiempo para la ecuacion CPU: %f\n\n",dwalltime() - timetick);
/*
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matC[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");
*/
    //--------------------------------cpu termina ------------------------------------



    for (i = 0; i < N*N; i++){
        matA[i] = i;
        matB[i] = i;
        matC[i] = 0;
    }

    //--------------------------------gpu out-place comienza ------------------------------------

	timetick = dwalltime();
    hipMemcpy(d_matA, matA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_matB, matB, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_matC, matC, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    ecuacion_kernel_outplace_p1<<<dimGrid, dimBlock>>>(d_matA, d_matAT,d_matB,d_matBT, N);
    hipDeviceSynchronize();
    ecuacion_kernel_outplace_p2<<<dimGrid, dimBlock>>>(d_matA, d_matB,d_matC,d_matAT,d_matBT, N);
    hipDeviceSynchronize();
    hipMemcpy(matC, d_matC, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU


    printf("Tiempo para la ecuacion out-place GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n\n",error);    
    
/*
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matC[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");
*/
    //--------------------------------gpu out-place termina ------------------------------------    

    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);
    hipFree(d_matAT);
    hipFree(d_matBT);


    for (i = 0; i < N*N; i++){
        matA[i] = i;
        matB[i] = i;
        matC[i] = 0;
    }

    hipMalloc((void **) &d_matA, numBytes);
    hipMalloc((void **) &d_matB, numBytes);
    hipMalloc((void **) &d_matC, numBytes);


    //--------------------------------gpu in-place comienza ------------------------------------

  

    timetick = dwalltime();
    hipMemcpy(d_matA, matA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_matB, matB, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    hipMemcpy(d_matC, matC, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
    kernel_transpuesta<<<dimGrid, dimBlock>>>(d_matA, N);
    hipDeviceSynchronize();
    kernel_sum_Matriz<<<dimGrid, dimBlock>>>(d_matA, d_matB,d_matC, N);
    hipDeviceSynchronize();
    kernel_transpuesta<<<dimGrid, dimBlock>>>(d_matA, N);
    hipDeviceSynchronize();
    kernel_mult_sum_matriz<<<dimGrid, dimBlock>>>(d_matA, d_matB,d_matC, N);
    hipDeviceSynchronize();

    hipMemcpy(matC, d_matC, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
	printf("Tiempo para la ecuacion in-place GPU: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n\n",error); 


    //--------------------------------gpu in-place termina ------------------------------------

  
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);
/*
  //imprime la matriz matC
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matC[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");

*/
    free(matA);
    free(matB);
    free(matC);
    return 0;
}
