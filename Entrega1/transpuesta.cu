#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

__global__ void kernel_transpuesta(double *m, int N){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int i = int((1 + sqrtf(1 + 8*tid)) / 2);
	int j = tid - (i*(i-1)/2); int aux;
	if ( (i<N) && (j<N) ){
		aux = m[i*N + j] ;
		m[i*N + j] = m[j*N + i];
		m[j*N + i] = aux;
	}
}


int main(int argc, char *argv[]){

    hipError_t error;
    unsigned int N = 8;
    unsigned long CUDA_BLK = 2, gridBlock;
    unsigned long numBytes = sizeof(double)*N*N;
    double *matA,*d_matA;
    unsigned int i,j;

	//inicializa variables para cpu
    matA = (double *)malloc(numBytes);
    for (i = 0; i < N*N; i++){
        matA[i] = i;
    }

  //inicializa variables para gpu
  hipMalloc((void **) &d_matA, numBytes);
  hipMemcpy(d_matA, matA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU

  gridBlock = (unsigned int)sqrt(N*N/CUDA_BLK/CUDA_BLK);
  dim3 dimBlock(CUDA_BLK,CUDA_BLK); // Bloque bidimencional de hilos (*cb* hilos)
  dim3 dimGrid(gridBlock,gridBlock); // Grid bidimencional (*ceil(n/cb)* bloques)

    kernel_transpuesta<<<dimGrid, dimBlock>>>(d_matA, N);
    hipDeviceSynchronize();
    kernel_transpuesta<<<dimGrid, dimBlock>>>(d_matA, N);
    hipDeviceSynchronize();
    error = hipGetLastError();
    printf("error: %d\n\n",error);

    //--------------------------------gpu in-place termina ------------------------------------

    hipMemcpy(matA, d_matA, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
  
  //imprime la matriz matA
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matA[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");


    hipFree(d_matA);
    free(matA);
    return 0;
}