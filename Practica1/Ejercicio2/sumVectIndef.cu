#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>

//134217728

//  Definición de nuestro kernel para función cuadradoV
__global__ void sumV_kernel_cuda(double *arrayA,double *arrayB ,   int n){

  unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (global_id < n)
    arrayA[global_id] = arrayA[global_id] + arrayB[global_id];

}

void checkparams(unsigned long *n, unsigned int *cb);
double dwalltime();


__global__ void sumV_kernel_cuda(double *d_vecA,double *d_vecB, long n, unsigned long dist){
    unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id + dist < n)
        d_vecA[global_id + dist] = d_vecA[global_id + dist] + d_vecB[global_id + dist];
}


int main(int argc, char *argv[]){
/*
    if (argc != 3){
        printf("Falta argumento: N, CUDA_BLK\n");
        return 0;
    }
    unsigned long N = atoi (argv[1]);
    unsigned int CUDA_BLK = atoi(argv[2]);*/
    unsigned long N = 107107840;
    unsigned int CUDA_BLK = 32;
    unsigned long max_N = N;
    checkparams(&max_N,&CUDA_BLK);
    double *vecA,*vecB,*d_vecA,*d_vecB,timetick;
    unsigned int i;
    hipError_t error;
    unsigned long numBytes =sizeof(double)*N ;
    struct hipDeviceProp_t capabilities;
    hipGetDeviceProperties (&capabilities, 0);

    vecA = (double *)malloc(numBytes);
    vecB = (double *)malloc(numBytes);

    for (i = 0; i < N; i++){
        vecA[i] = i;
        vecB[i] = i;
    }

  hipMalloc((void **) &d_vecA, numBytes);
  hipMalloc((void **) &d_vecB, numBytes);
  hipMemcpy(d_vecA, vecA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  hipMemcpy(d_vecB, vecB, numBytes, hipMemcpyHostToDevice); // CPU -> GPU

  // Bloque unidimensional de hilos (*cb* hilos)
  dim3 dimBlock(32);
  // Grid unidimensional (*ceil(n/cb)* bloques)
  dim3 dimGrid((max_N + dimBlock.x - 1) / dimBlock.x);

     long aux_N = N;
	timetick = dwalltime();
    int rep = 0;
    while(aux_N > 0){
        printf("%lu\n",aux_N);
        sumV_kernel_cuda<<<dimGrid, dimBlock>>>(d_vecA, d_vecB, N, max_N*rep);
        aux_N = aux_N - max_N;
        rep++;
    }
    hipDeviceSynchronize();

  printf("-> Tiempo de ejecucion en GPU %f\n", dwalltime() - timetick);
  error = hipGetLastError();

  // Movemos resultado: GPU -> CPU
  timetick = dwalltime();

  hipMemcpy(vecA, d_vecA, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
  printf("-> Tiempo de copia GPU ==>> CPU %f\n", dwalltime() - timetick);

    for(i= 0; i < 20; i++){
        printf("%f|",vecA[i]);
    }
	printf("\n");
    printf("error code: %d\n",error);
    printf("\n%lu||||%lu\n",(max_N + dimBlock.x - 1) / dimBlock.x,CUDA_BLK);

    hipFree (vecA);
    hipFree (vecB);
    free(vecA);
    free(vecB);
    return 0;
}

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

void checkparams(unsigned long *n, unsigned int *cb){
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %lu (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}

