
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>

//134217728

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

int main(int argc, char *argv[]){

    if (argc == 1){
        printf("Falta argumento: N\n");
        return 0;
    }
    unsigned int N = atoi (argv[1]);
    double *vecA,*vecB,timetick;
    unsigned int i;

    vecA = (double *)malloc(sizeof(double)*N);
    vecB = (double *)malloc(sizeof(double)*N);

    for (i = 0; i < N; i++){
        vecA[i] = i;
        vecB[i] = i;
    }

	timetick = dwalltime();
    for (i = 0; i < N; i++){
		vecA[i] = vecA[i] + vecB[i];
	}
	printf("Tiempo para sumar los vectores: %f\n",dwalltime() - timetick);

    for(i= 0; i < 20; i++){
        printf("%f|",vecA[i]);
    }
	printf("\n");

    return 0;
}