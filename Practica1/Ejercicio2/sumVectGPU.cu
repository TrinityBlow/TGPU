
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>

//134217728

//  Definición de nuestro kernel para función cuadradoV
__global__ void sumV_kernel_cuda(double *arrayA,double *arrayB ,   int n){

  unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (global_id < n)
    arrayA[global_id] = arrayA[global_id] + arrayB[global_id];

}

void checkparams(unsigned long *n, unsigned int *cb);
double dwalltime();


int main(int argc, char *argv[]){

    if (argc == 2){
        printf("Falta argumento: N, CUDA_BLK\n");
        return 0;
    }
    unsigned long N = atoi (argv[1]);
    unsigned int CUDA_BLK = atoi(argv[2]);
    checkparams(&N,&CUDA_BLK);
    double *vecA,*vecB,timetick;
    unsigned int i;
    hipError_t error;
    
    vecA = (double *)malloc(sizeof(double)*N);
    vecB = (double *)malloc(sizeof(double)*N);

    for (i = 0; i < N; i++){
        vecA[i] = i;
        vecB[i] = i;
    }

	timetick = dwalltime();



	printf("Tiempo para sumar los vectoresGPU: %f\n",dwalltime() - timetick);

    for(i= 0; i < 20; i++){
        printf("%f|",vecA[i]);
    }
	printf("\n");

    return 0;
}

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

void checkparams(unsigned long *n, unsigned int *cb){
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %lu (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}

