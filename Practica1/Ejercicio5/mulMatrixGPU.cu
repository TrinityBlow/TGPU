#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>

//134217728

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

__global__ void mulM_kernel_cuda(double *d_matA,double *d_matB,double *d_matC, unsigned long n){    
    unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int k,distA = global_id / n, distB = global_id % n;
    if (global_id < n*n){
        for(k = 0; k < n ;k++){
            d_matC[global_id] += d_matA[distA*n+k] * d_matB[distB*n+k];
        }
    }
}


void checkparams(unsigned long *n, unsigned int *cb);

int main(int argc, char *argv[]){

        if (argc != 2){
            printf("Falta argumento: N\n");
            return 0;
        }
    hipError_t error;

    unsigned long N = atoi (argv[1]),tam_tot = N*N;
    unsigned int CUDA_BLK = 32;
    unsigned long numBytes = sizeof(double)*tam_tot;
    checkparams(&tam_tot,&CUDA_BLK);
    double *matA,*matB,*matC,*d_matA,*d_matB,*d_matC,timetick;
    unsigned int i,j;


    matA = (double *)malloc(numBytes);
    matB = (double *)malloc(numBytes);
    matC = (double *)malloc(numBytes);

    for (i = 0; i < tam_tot; i++){
        matA[i] = 2;
        matB[i] = 3;
        matC[i] = 0;
    }

  hipMalloc((void **) &d_matA, numBytes);
  hipMalloc((void **) &d_matB, numBytes);
  hipMalloc((void **) &d_matC, numBytes);
  hipMemcpy(d_matA, matA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  hipMemcpy(d_matB, matB, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  hipMemcpy(d_matC, matC, numBytes, hipMemcpyHostToDevice); // CPU -> GPU


  // Bloque unidimensional de hilos (*cb* hilos)
  dim3 dimBlock(CUDA_BLK);
  // Grid unidimensional (*ceil(n/cb)* bloques)
  dim3 dimGrid((tam_tot + dimBlock.x - 1) / dimBlock.x);


    

	timetick = dwalltime();
    mulM_kernel_cuda<<<dimGrid, dimBlock>>>(d_matA, d_matB,d_matC, N);
    hipDeviceSynchronize();
	printf("Tiempo para sumar las matrices: %f\n",dwalltime() - timetick);

  hipMemcpy(matC, d_matC, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
  /*
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matC[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");
    */

printf("%lu|||||||\n",CUDA_BLK*(tam_tot + dimBlock.x - 1) / dimBlock.x);
    error = hipGetLastError();
    printf("error: %d\n",error);
    printf("%.2lf\n",matC[0]);
    printf("%.2lf\n",matC[N*N-1]);

    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);
    free(matA);
    free(matB);
    free(matC);
    return 0;
}

void checkparams(unsigned long *n, unsigned int *cb){
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %lu (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}
