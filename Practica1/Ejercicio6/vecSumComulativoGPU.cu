#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>

//134217728

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}


__global__ void vecSum_kernel_cuda(double *d_vecA,double *d_result,unsigned long rep,unsigned long n){    
    unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    double aux_sum = 0;
    __shared__ double aux_result;
    __shared__ int esperar[1024];
    __shared__ int termino;

    if (global_id == 0){
        esperar[global_id] = 0;
    printf("%d|%d\n",global_id,esperar[global_id]);
    } else {
        esperar[global_id] = 1;
    printf("%d|%d\n",global_id,esperar[global_id]);
    }
    if(global_id == 0){
        aux_result = 0;
        termino = 1;
    }
    __syncthreads();
    if (global_id < n){
        for (i = 0; i < rep; i++){
            aux_sum = aux_sum + d_vecA[global_id*rep+i];
        }
    }
   while (esperar[global_id]){
        __syncthreads();
    }
    aux_result = aux_result + aux_sum;
    if(global_id == 3){
        *d_result = aux_result;
        termino = 0;
    } else{
        esperar[global_id+1] = 0;
    }
    __syncthreads();
    while(termino){
        __syncthreads();
    }
}

void checkparams(unsigned long *n, unsigned int *cb);
void checkparamsB(unsigned long *n, unsigned int *cb);


int main(int argc, char *argv[]){

    if (argc != 2){
        printf("Falta argumento: N\n");
        return 0;
    }


    unsigned long N = atoi (argv[1]);
    double *vecA,*result,timetick;
    unsigned int i;

    hipError_t error;
    unsigned int CUDA_BLK = 4;
   // checkparamsB(&N,&CUDA_BLK);
    unsigned long numBytes = sizeof(double)*N;
    double *d_vecA,*d_result;


    vecA = (double *)malloc(numBytes); 
    result = (double *)malloc(sizeof(double)); 
    *result = 0;   
    for (i = 0; i < N; i++){
        vecA[i] = i;
    }
  
    hipMalloc((void **) &d_vecA, numBytes);  
    hipMalloc((void **) &d_result, sizeof(double));  
    hipMemcpy(d_vecA, vecA, numBytes, hipMemcpyHostToDevice);


    // Bloque unidimensional de hilos (*cb* hilos)
    dim3 dimBlock(CUDA_BLK);
    // Grid unidimensional (*ceil(n/cb)* bloques)
    dim3 dimGrid(1);



    timetick = dwalltime();
    vecSum_kernel_cuda<<<dimGrid, dimBlock>>>(d_vecA,d_result,N/CUDA_BLK,CUDA_BLK);
    hipDeviceSynchronize();
    printf("Tiempo para sumar las matrices: %f\n",dwalltime() - timetick);

    hipMemcpy(result, d_result, sizeof(double), hipMemcpyDeviceToHost); // GPU -> CPU



    printf("%d|||||||\n",CUDA_BLK*1);
    error = hipGetLastError();
    printf("error: %d\n",error);
    printf("%f\n",*result);

    hipFree(d_vecA);
    hipFree(d_result);
    free(vecA);
    free(result);

    return 0;
}


void checkparamsB(unsigned long *n, unsigned int *cb){

    struct hipDeviceProp_t capabilities;
    hipGetDeviceProperties (&capabilities, 0);
    *cb = capabilities.maxThreadsDim[0];
    printf("%d\n",*cb);
}

void checkparams(unsigned long *n, unsigned int *cb){
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %lu (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}