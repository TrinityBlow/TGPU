#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>

//134217728

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}


__global__ void vecSum_kernel_cuda(double *d_vecA,double *d_result,unsigned long dist,unsigned long n){    
    unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < n){
        d_vecA[global_id*dist] = d_vecA[global_id*dist] + d_vecA[global_id*dist+dist / 2];
    }
}

void checkparams(unsigned long *n, unsigned int *cb);
void checkparamsB(unsigned long *n, unsigned int *cb);


int main(int argc, char *argv[]){

    if (argc != 2){
        printf("Falta argumento: N\n");
        return 0;
    }


    unsigned long N = atoi (argv[1]);
    double *vecA,*result,timetick;
    unsigned int i;

    hipError_t error;
    unsigned int CUDA_BLK = 32;
   // checkparamsB(&N,&CUDA_BLK);
    unsigned long numBytes = sizeof(double)*N;
    double *d_vecA,*d_result;


    vecA = (double *)malloc(numBytes); 
    result = (double *)malloc(sizeof(double)); 
    *result = 0;   
    for (i = 0; i < N; i++){
        vecA[i] = i;
    }
  
    hipMalloc((void **) &d_vecA, numBytes);  
    hipMalloc((void **) &d_result, sizeof(double));  
    hipMemcpy(d_vecA, vecA, numBytes, hipMemcpyHostToDevice);


    // Bloque unidimensional de hilos (*cb* hilos)
    dim3 dimBlock(CUDA_BLK);
    // Grid unidimensional (*ceil(n/cb)* bloques)



    timetick = dwalltime();
    for(i = 2; i <= N ;i *= 2){
        dim3 dimGrid((N / i + dimBlock.x - 1) / dimBlock.x);
        vecSum_kernel_cuda<<<dimGrid, dimBlock>>>(d_vecA,d_result,i,N/i);
        hipDeviceSynchronize();
    }
    printf("Tiempo para sumar las matrices: %f\n",dwalltime() - timetick);
    error = hipGetLastError();
    printf("error: %d\n",error);

    hipMemcpy(result, d_vecA, sizeof(double), hipMemcpyDeviceToHost); // GPU -> CPU



    printf("%f\n",*result);

    hipFree(d_vecA);
    hipFree(d_result);
    free(vecA);
    free(result);

    return 0;
}


void checkparamsB(unsigned long *n, unsigned int *cb){

    struct hipDeviceProp_t capabilities;
    hipGetDeviceProperties (&capabilities, 0);
    *cb = capabilities.maxThreadsDim[0];
    printf("%d\n",*cb);
}

void checkparams(unsigned long *n, unsigned int *cb){
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %lu (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}