#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <math.h>

//134217728

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

__global__ void transpuesta_out_place (double *mt, double *m,unsigned int N)
{
    int i = blockIdx.y * blockDim.y +  threadIdx.y;
    int j = blockIdx.x * blockDim.x +  threadIdx.x;
    if( (i<N*N) && (j<N*N) ){
        mt [j*N + i] = m[i*N + j];
    }
    
}


void checkparams(unsigned long *n, unsigned int *cb);

int main(int argc, char *argv[]){

        if (argc != 2){
            printf("Falta argumento: N\n");
            return 0;
        }
    hipError_t error;

    unsigned int N = atoi (argv[1]),tam_tot = N*N;
    unsigned int CUDA_BLK = 2, gridBlock;
    unsigned long numBytes = sizeof(double)*tam_tot;
    double *matA,*matB,*d_matA,*d_matB,timetick;
    unsigned int i,j;


    matA = (double *)malloc(numBytes);
    matB = (double *)malloc(numBytes);

    for (i = 0; i < tam_tot; i++){
        matA[i] = i;
        matB[i] = 0;
    }

  hipMalloc((void **) &d_matA, numBytes);
  hipMalloc((void **) &d_matB, numBytes);
  hipMemcpy(d_matA, matA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  hipMemcpy(d_matB, matB, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  gridBlock = (unsigned int)sqrt(N*N/CUDA_BLK/CUDA_BLK);


  printf("%u||%u||%u||\n",CUDA_BLK,gridBlock,N);
  printf("dimBlockSize:%u\ndimGridSize:%u\ntotalMatriz:%u\n",CUDA_BLK*CUDA_BLK,gridBlock*gridBlock,N*N);
  // Bloque unidimensional de hilos (*cb* hilos)
  dim3 dimBlock(CUDA_BLK,CUDA_BLK);
  // Grid unidimensional (*ceil(n/cb)* bloques)
  dim3 dimGrid(gridBlock,gridBlock);


    

	timetick = dwalltime();
    transpuesta_out_place<<<dimGrid, dimBlock>>>(d_matB, d_matA, N);
    hipDeviceSynchronize();
	printf("Tiempo para sumar las matrices: %f\n",dwalltime() - timetick);

  hipMemcpy(matB, d_matB, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
  /*
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matC[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");
    */

printf("%u|||||||\n",CUDA_BLK*(tam_tot + dimBlock.x - 1) / dimBlock.x);
    error = hipGetLastError();
    printf("error: %d\n",error);
    printf("%.2lf\n",matB[1]);
    printf("%.2lf\n",matB[N*N-2]);

    hipFree(d_matA);
    hipFree(d_matB);
    free(matA);
    free(matB);
    return 0;
}

void checkparams(unsigned long *n, unsigned int *cb){
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %lu (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}
