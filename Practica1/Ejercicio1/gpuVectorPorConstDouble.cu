#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <hip/driver_types.h>


void checkparams(unsigned long *n, unsigned int *cb);

double dwalltime(){
        double sec;
        struct timeval tv;

        gettimeofday(&tv,NULL);
        sec = tv.tv_sec + tv.tv_usec/1000000.0;
        return sec;
}

typedef double basetype;  // Tipo para elementos: double
#define labelelem    "doubles"


const unsigned long N = 134217728 * 4 ;    // Número predeterminado de elementos en los vectores

const int CUDA_BLK = 64;  // Tamaño predeterminado de bloque de hilos CUDA

basetype C = 10;

//  Definición de nuestro kernel para función cuadradoV
__global__ void constV_kernel_cuda(basetype *const arrayV,   const int n, basetype c){

  unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (global_id < n)
    arrayV[global_id] = arrayV[global_id]*c;

}



int main(int argc, char *argv[]){
 
  // Número de elementos del vector (predeterminado: N 1048576)
  unsigned long n = (argc > 1)?atoi (argv[1]):N;
  // Número de hilos en cada bloque CUDA (predeterminado: CUDA_BLK 64)
  unsigned int cb = (argc > 2)?atoi (argv[2]):CUDA_BLK;
  checkparams(&n, &cb);
  unsigned int numBytes = n * sizeof(basetype);
  unsigned int i;
  basetype *vectorV = (basetype *) malloc(numBytes);
  // Reservamos memoria global del device (GPU) para el array y lo copiamos
  basetype *cV;
  double timetick;
  hipError_t error;

  for(i = 0; i < n; i++) {
    vectorV[i] = (basetype)i;
  }

  hipMalloc((void **) &cV, numBytes);
  hipMemcpy(cV, vectorV, numBytes, hipMemcpyHostToDevice); // CPU -> GPU

  // Bloque unidimensional de hilos (*cb* hilos)
  dim3 dimBlock(cb);

  // Grid unidimensional (*ceil(n/cb)* bloques)
  dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x);

  timetick = dwalltime();
  constV_kernel_cuda<<<dimGrid, dimBlock>>>(cV, n, C);
  hipDeviceSynchronize();
  printf("-> Tiempo de ejecucion en GPU %f\n", dwalltime() - timetick);
  error = hipGetLastError();

  // Movemos resultado: GPU -> CPU
  timetick = dwalltime();
  hipMemcpy(vectorV, cV, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
  printf("-> Tiempo de copia GPU ==>> CPU %f\n", dwalltime() - timetick);

  for (i = 0; i < 20; i++){
    printf("%f|",vectorV[i]);
  }
 printf("\n");
 unsigned long temp = n-1;
  for (i = 0; i < 20; i++){
    printf("%f|",vectorV[temp]);
	temp--;
  }
  printf("\n%lu",n);
  printf("\n");
  printf("%d\n",error);

  // Liberamos memoria global del device utilizada
  hipFree (cV);
  free(vectorV);
}


//  Función que ajusta el número de hilos, de bloques, y de bloques por hilo 
//  de acuerdo a las restricciones de la GPU
void checkparams(unsigned long *n, unsigned int *cb){
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %lu (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}
