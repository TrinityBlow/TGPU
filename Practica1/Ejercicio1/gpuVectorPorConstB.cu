#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <hip/driver_types.h>


void checkparams(unsigned long *n, unsigned int *cb);
double dwalltime(){
        double sec;
        struct timeval tv;

        gettimeofday(&tv,NULL);
        sec = tv.tv_sec + tv.tv_usec/1000000.0;
        return sec;
}

#define _INT_

#ifdef _INT_
typedef int basetype;     // Tipo para elementos: int 
#define labelelem    "ints"
#elif _DOUBLE_
typedef double basetype;  // Tipo para elementos: double
#define labelelem    "doubles"
#else
typedef float basetype;   // Tipo para elementos: float PREDETERMINADO
#define labelelem    "floats"
#endif


__constant__ unsigned long d_n ;
__constant__ basetype d_C = 10;


const unsigned long N = 134217728 * 4 ;    // Número predeterminado de elementos en los vectores

const int CUDA_BLK = 64;  // Tamaño predeterminado de bloque de hilos CUDA


//  Definición de nuestro kernel para función cuadradoV
__global__ void constV_kernel_cuda(basetype *const cV){

  unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (global_id < d_n)
    cV[global_id] = cV[global_id]*d_C;

}



int main(int argc, char *argv[]){
 
  // Número de elementos del vector (predeterminado: N 1048576)
  unsigned long n = (argc > 1)?atoi (argv[1]):N;
  // Número de hilos en cada bloque CUDA (predeterminado: CUDA_BLK 64)
  unsigned int cb = (argc > 2)?atoi (argv[2]):CUDA_BLK;
  checkparams(&n, &cb);
  unsigned int numBytes = n * sizeof(basetype);
  unsigned int i;
  basetype *vectorV = (basetype *) malloc(numBytes);
  // Reservamos memoria global del device (GPU) para el array y lo copiamos
  double timetick;
  hipError_t error;
	basetype *cV;

  for(i = 0; i < n; i++) {
    vectorV[i] = (basetype)i;
  }

  hipMalloc((void **) &cV, numBytes);
  hipMemcpy(cV, vectorV, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  hipMemcpyToSymbol(HIP_SYMBOL(d_n), &n, sizeof(unsigned long) ); // CPU -> GPU

  // Bloque unidimensional de hilos (*cb* hilos)
  dim3 dimBlock(cb);

  // Grid unidimensional (*ceil(n/cb)* bloques)
  dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x);

  timetick = dwalltime();
  constV_kernel_cuda<<<dimGrid, dimBlock>>>(cV);
  hipDeviceSynchronize();
  printf("-> Tiempo de ejecucion en GPU %f\n", dwalltime() - timetick);
  error = hipGetLastError();

  // Movemos resultado: GPU -> CPU
  timetick = dwalltime();
  hipMemcpy(vectorV, cV, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
  printf("-> Tiempo de copia GPU ==>> CPU %f\n", dwalltime() - timetick);

  for (i = 0; i < 20; i++){
    printf("%d|",vectorV[i]);
  }
 printf("\n");
 unsigned long temp = n-1;
  for (i = 0; i < 20; i++){
    printf("%d|",vectorV[temp]);
	temp--;
  }
  printf("\n%lu",n);
  printf("\n");
  printf("%d\n",error);

  // Liberamos memoria global del device utilizada
  hipFree (cV);
  free(vectorV);
}


//  Función que ajusta el número de hilos, de bloques, y de bloques por hilo 
//  de acuerdo a las restricciones de la GPU
void checkparams(unsigned long *n, unsigned int *cb){
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %lu (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}
