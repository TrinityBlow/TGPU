#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>

//134217728

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

__global__ void sumM_kernel_cuda(double *d_matA,double *d_matB, unsigned long n){    
    unsigned long int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    if (global_id < n)
        for(i = 0; i < n;i++){
            d_matA[global_id*n+i] = d_matA[global_id*n+i] + d_matB[global_id*n+i];
        }
}


void checkparams(unsigned long *n, unsigned int *cb);

int main(int argc, char *argv[]){

    if (argc != 2){
        printf("Falta argumento: N\n");
        return 0;
    }
    unsigned long N = atoi (argv[1]),tam_tot = N*N;
    unsigned int CUDA_BLK = 4;
    unsigned long numBytes = sizeof(double)*tam_tot;
    checkparams(&tam_tot,&CUDA_BLK);
    double *matA,*matB,*d_matA,*d_matB,timetick;
    unsigned int i,j;
    matA = (double *)malloc(numBytes);
    matB = (double *)malloc(numBytes);

    for (i = 0; i < tam_tot; i++){
        matA[i] = i;
        matB[i] = i;
    }


  hipMalloc((void **) &d_matA, numBytes);
  hipMalloc((void **) &d_matB, numBytes);
  hipMemcpy(d_matA, matA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  hipMemcpy(d_matB, matB, numBytes, hipMemcpyHostToDevice); // CPU -> GPU

  // Bloque unidimensional de hilos (*cb* hilos)
  dim3 dimBlock(CUDA_BLK);
  // Grid unidimensional (*ceil(n/cb)* bloques)
  dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);
    hipError_t error;

	timetick = dwalltime();
    sumM_kernel_cuda<<<dimGrid, dimBlock>>>(d_matA, d_matB, N);
    hipDeviceSynchronize();
	printf("Tiempo para sumar las matrices: %f\n",dwalltime() - timetick);


error = hipGetLastError();
printf("error: %d\n",error);
  hipMemcpy(matA, d_matA, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    /*
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matA[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");*/
    
    hipFree(d_matA);
    hipFree(d_matB);
    free(matA);
    free(matB);
    return 0;
}

void checkparams(unsigned long *n, unsigned int *cb){
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %lu (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}