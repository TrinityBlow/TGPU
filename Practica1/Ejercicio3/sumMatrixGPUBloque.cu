#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <sys/resource.h>

//134217728

double dwalltime(){
	double sec;
	struct timeval tv;

	gettimeofday(&tv,NULL);
	sec = tv.tv_sec + tv.tv_usec/1000000.0;
	return sec;
}

__global__ void sumM_kernel_cuda(double *d_matA,double *d_matB, unsigned long n){    
    int k;
    int distA = blockIdx.y * blockDim.y + threadIdx.y; //i
    int distB = blockIdx.x * blockDim.x + threadIdx.x; //j
    if (distA*n+distB < n*n){
        d_matA[distA*n+distB] += d_matB[distB*n+k];
    }
}


void checkparams(unsigned long *n, unsigned int *cb);

int main(int argc, char *argv[]){

    if (argc != 2){
        printf("Falta argumento: N\n");
        return 0;
    }
    unsigned long N = atoi (argv[1]),tam_tot = N*N;
    unsigned int CUDA_BLK = 8, gridBlock;
    unsigned long numBytes = sizeof(double)*tam_tot;
    double *matA,*matB,*d_matA,*d_matB,timetick;
    unsigned int i,j;
    matA = (double *)malloc(numBytes);
    matB = (double *)malloc(numBytes);

    for (i = 0; i < tam_tot; i++){
        matA[i] = i;
        matB[i] = i;
    }


  hipMalloc((void **) &d_matA, numBytes);
  hipMalloc((void **) &d_matB, numBytes);
  hipMemcpy(d_matA, matA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU
  hipMemcpy(d_matB, matB, numBytes, hipMemcpyHostToDevice); // CPU -> GPU

  // Bloque unidimensional de hilos (*cb* hilos)
  dim3 dimBlock(CUDA_BLK);
  // Grid unidimensional (*ceil(n/cb)* bloques)
  dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);
    hipError_t error;

	timetick = dwalltime();
    sumM_kernel_cuda<<<dimGrid, dimBlock>>>(d_matA, d_matB, N);
    hipDeviceSynchronize();
	printf("Tiempo para sumar las matrices: %f\n",dwalltime() - timetick);


error = hipGetLastError();
printf("error: %d\n",error);
  hipMemcpy(matA, d_matA, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU
    /*
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%f|",matA[i*N+j]);
        }
        printf("\n");
    }
	printf("\n");*/
    
    hipFree(d_matA);
    hipFree(d_matB);
    free(matA);
    free(matB);
    return 0;
}

void checkparams(unsigned long *n, unsigned int *cb){
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %lu (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}